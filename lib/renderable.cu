#include "hip/hip_runtime.h"
#include <cassert>
#include "renderable.hpp"

////////////////////////////////////////////////////////////////////////////////

template <typename R>
__device__ void storeAxes(const uint32_t index, const uint32_t tile,
                          const View& v, const Tiles& tiles, const Tape& tape,
                          R* const __restrict__ lower,
                          R* const __restrict__ upper)
{
   // Prepopulate axis values
    const float x = tile / tiles.per_side;
    const float y = tile % tiles.per_side;

    Interval vs[3];
    const Interval X = {x / tiles.per_side, (x + 1) / tiles.per_side};
    vs[0].lower = 2.0f * (X.lower - 0.5f) * v.scale - v.center[0];
    vs[0].upper = 2.0f * (X.upper - 0.5f) * v.scale - v.center[0];

    const Interval Y = {y / tiles.per_side, (y + 1) / tiles.per_side};
    vs[1].lower = 2.0f * (Y.lower - 0.5f) * v.scale - v.center[1];
    vs[1].upper = 2.0f * (Y.upper - 0.5f) * v.scale - v.center[1];

    vs[2].lower = 0.0f;
    vs[2].upper = 0.0f;

    for (unsigned i=0; i < 3; ++i) {
        if (tape.axes.reg[i] != UINT16_MAX) {
            lower[tape.axes.reg[i]][index] = vs[i].lower;
            upper[tape.axes.reg[i]][index] = vs[i].upper;
        }
    }
}

template <typename A, typename B, typename C>
__device__ inline Interval intervalOp(uint8_t op, A lhs, B rhs, C*& choices)
{
    using namespace libfive::Opcode;
    switch (op) {
        case OP_SQUARE: return square(lhs);
        case OP_SQRT: return sqrt(lhs);
        case OP_NEG: return -lhs;
        // Skipping transcendental functions for now

        case OP_ADD: return lhs + rhs;
        case OP_MUL: return lhs * rhs;
        case OP_DIV: return lhs / rhs;
        case OP_MIN: if (upper(lhs) < lower(rhs)) {
                         (*choices++)[threadIdx.x] = 1;
                         return lhs;
                     } else if (upper(rhs) < lower(lhs)) {
                         (*choices++)[threadIdx.x] = 2;
                         return rhs;
                     } else {
                         (*choices++)[threadIdx.x] = 0;
                         return min(lhs, rhs);
                     }
        case OP_MAX: if (lower(lhs) > upper(rhs)) {
                         (*choices++)[threadIdx.x] = 1;
                         return lhs;
                     } else if (lower(rhs) > upper(lhs)) {
                         (*choices++)[threadIdx.x] = 2;
                         return rhs;
                     } else {
                         (*choices++)[threadIdx.x] = 0;
                         return max(lhs, rhs);
                     }
        case OP_SUB: return lhs - rhs;

        // Skipping various hard functions here
        default: break;
    }
    return {0.0f, 0.0f};
}

////////////////////////////////////////////////////////////////////////////////

TileRenderer::TileRenderer(const Tape& tape, Image& image)
    : tape(tape), image(image),
      tiles(image.size_px, LIBFIVE_CUDA_TILE_SIZE_PX),

      regs_lower(CUDA_MALLOC(Registers, LIBFIVE_CUDA_TILE_BLOCKS *
                                        tape.num_regs * 2)),
      regs_upper(regs_lower + LIBFIVE_CUDA_TILE_BLOCKS * tape.num_regs),
      active(CUDA_MALLOC(ActiveArray, LIBFIVE_CUDA_TILE_BLOCKS *
                                      tape.num_regs)),
      choices(tape.num_csg_choices ?
              CUDA_MALLOC(ChoiceArray,
                LIBFIVE_CUDA_TILE_BLOCKS * tape.num_csg_choices)
              : nullptr)
{
    // Nothing to do here
}

TileRenderer::~TileRenderer()
{
    CHECK(hipFree(regs_lower));
    CHECK(hipFree(active));
    CHECK(hipFree(choices));
}

__device__
void TileRenderer::check(const uint32_t tile, const View& v)
{
    auto regs_lower = this->regs_lower + tape.num_regs * blockIdx.x;
    auto regs_upper = this->regs_upper + tape.num_regs * blockIdx.x;
    storeAxes(threadIdx.x, tile, v, tiles, tape, regs_lower, regs_upper);

    // Unpack a 1D offset into the data arrays
    auto choices = this->choices + tape.num_csg_choices * blockIdx.x;

    const Clause* __restrict__ clause_ptr = &tape[0];
    const float* __restrict__ constant_ptr = &tape.constant(0);
    const auto num_clauses = tape.num_clauses;

    // We copy a chunk of the tape from constant to shared memory
    constexpr unsigned SHARED_CLAUSE_SIZE = LIBFIVE_CUDA_TILE_THREADS;
    __shared__ Clause clauses[SHARED_CLAUSE_SIZE];
    __shared__ float constant_lhs[SHARED_CLAUSE_SIZE];
    __shared__ float constant_rhs[SHARED_CLAUSE_SIZE];

    for (uint32_t i=0; i < num_clauses; ++i) {
        using namespace libfive::Opcode;

        if ((i % SHARED_CLAUSE_SIZE) == 0) {
            __syncthreads();
            if (i + threadIdx.x < num_clauses) {
                const Clause c = clause_ptr[i + threadIdx.x];
                if (c.banks & 1) {
                    constant_lhs[threadIdx.x] = constant_ptr[c.lhs];
                }
                if (c.banks & 2) {
                    constant_rhs[threadIdx.x] = constant_ptr[c.rhs];
                }
                clauses[threadIdx.x] = c;
            }
            __syncthreads();
        }

        // Skip unused tiles
        if (tile == UINT32_MAX) {
            continue;
        }

        const Clause c = clauses[i % SHARED_CLAUSE_SIZE];
        // All clauses must have at least one argument, since constants
        // and VAR_X/Y/Z are handled separately.
        Interval lhs;
        if (c.banks & 1) {
            const float f = constant_lhs[i % SHARED_CLAUSE_SIZE];
            lhs.lower = f;
            lhs.upper = f;
        } else {
            lhs.lower = regs_lower[c.lhs][threadIdx.x];
            lhs.upper = regs_upper[c.lhs][threadIdx.x];
        }

        Interval rhs;
        if (c.banks & 2) {
            const float f = constant_rhs[i % SHARED_CLAUSE_SIZE];
            rhs.lower = f;
            rhs.upper = f;
        } else if (c.opcode >= OP_ADD) {
            rhs.lower = regs_lower[c.rhs][threadIdx.x];
            rhs.upper = regs_upper[c.rhs][threadIdx.x];
        }

        Interval out = intervalOp(c.opcode, lhs, rhs, choices);

        regs_lower[c.out][threadIdx.x] = out.lower;
        regs_upper[c.out][threadIdx.x] = out.upper;
    }

    uint32_t build_tape_tile = UINT32_MAX;
    if (tile != UINT32_MAX) {
        // Copy output to standard register before exiting
        const Clause c = clause_ptr[num_clauses - 1];
        const Interval result = {regs_lower[c.out][threadIdx.x],
                                 regs_upper[c.out][threadIdx.x]};

        // If this tile is unambiguously filled, then mark it at the end
        // of the tiles list
        if (result.upper < 0.0f) {
            tiles.insert_filled(tile);
        }

        // If the tile is ambiguous, then record it as needing further refinement
        else if ((result.lower <= 0.0f && result.upper >= 0.0f)
                || isnan(result.lower)
                || isnan(result.upper))
        {
            tiles.insert_active(tile);
            build_tape_tile = tile;
        }
    }

    ////////////////////////////////////////////////////////////////////////////
    // Now, we build a tape for this tile (if it's active).  If it isn't active,
    // then we use the thread to help copy stuff to shared memory, but don't
    // write any tape data out.

    // Pick a subset of the active array to use for this block
    auto active = this->active + blockIdx.x * tape.num_regs;

    for (uint32_t r=0; r < tape.num_regs; ++r) {
        active[r][threadIdx.x] = false;
    }

    // Mark the root of the tree as true
    active[tape[num_clauses - 1].out][threadIdx.x] = true;

    uint32_t subtape_index = 0;
    uint32_t s = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;

    // Claim a subtape to populate
    if (build_tape_tile != UINT32_MAX) {
        subtape_index = atomicAdd(&tiles.num_subtapes, 1);
        assert(subtape_index < LIBFIVE_CUDA_NUM_SUBTAPES);

        // Since we're reversing the tape, this is going to be the
        // end of the linked list (i.e. next = 0)
        tiles.subtapes.next[subtape_index] = 0;
    }

    // Walk from the root of the tape downwards
    Clause* __restrict__ out = tiles.subtapes.data[subtape_index];

    for (uint32_t i=0; i < num_clauses; i++) {
        using namespace libfive::Opcode;

        if ((i % SHARED_CLAUSE_SIZE) == 0) {
            __syncthreads();
            const uint32_t j = num_clauses - i - 1 - threadIdx.x;
            if (j < num_clauses) {
                clauses[SHARED_CLAUSE_SIZE - threadIdx.x - 1] = clause_ptr[j];
            }
            __syncthreads();
        }

        // Skip dummy tiles which don't actually do things
        if (build_tape_tile == UINT32_MAX) {
            continue;
        }
        Clause c = clauses[SHARED_CLAUSE_SIZE - (i % SHARED_CLAUSE_SIZE) - 1];

        if (active[c.out][threadIdx.x]) {
            active[c.out][threadIdx.x] = false;
            if (c.opcode == OP_MIN || c.opcode == OP_MAX) {
                const uint8_t choice = (*(--choices))[threadIdx.x];
                if (choice == 1) {
                    if (!(c.banks & 1)) {
                        active[c.lhs][threadIdx.x] = true;
                        if (c.lhs == c.out) {
                            continue;
                        }
                        c.rhs = c.lhs;
                        c.banks = 0;
                    } else {
                        c.rhs = c.lhs;
                        c.banks = 3;
                    }
                } else if (choice == 2) {
                    if (!(c.banks & 2)) {
                        active[c.rhs][threadIdx.x] = true;
                        if (c.rhs == c.out) {
                            continue;
                        }
                        c.lhs = c.rhs;
                        c.banks = 0;
                    } else {
                        c.lhs = c.rhs;
                        c.banks = 3;
                    }
                } else if (choice == 0) {
                    if (!(c.banks & 1)) {
                        active[c.lhs][threadIdx.x] = true;
                    }
                    if (!(c.banks & 2)) {
                        active[c.rhs][threadIdx.x] = true;
                    }
                } else {
                    assert(false);
                }
            } else {
                if (!(c.banks & 1)) {
                    active[c.lhs][threadIdx.x] = true;
                }
                if (c.opcode >= OP_ADD && !(c.banks & 2)) {
                    active[c.rhs][threadIdx.x] = true;
                }
            }

            // Allocate a new subtape and begin writing to it
            if (s == 0) {
                auto next_subtape_index = atomicAdd(&tiles.num_subtapes, 1);
                tiles.subtapes.start[subtape_index] = 0;
                tiles.subtapes.next[next_subtape_index] = subtape_index;

                subtape_index = next_subtape_index;
                s = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
                out = tiles.subtapes.data[subtape_index];
            }
            out[--s] = c;
        } else if (c.opcode == OP_MIN || c.opcode == OP_MAX) {
            --choices;
        }
    }

    if (build_tape_tile != UINT32_MAX) {
        // The last subtape may not be completely filled
        tiles.subtapes.start[subtape_index] = s;
        tiles.head(build_tape_tile) = subtape_index;
    }
}

__global__ void TileRenderer_check(TileRenderer* r,
                                   const uint32_t offset,
                                   View v)
{
    // This should be a 1D kernel
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t tile = threadIdx.x + blockIdx.x * blockDim.x + offset;
    r->check(tile < r->tiles.total ? tile : UINT32_MAX, v);
}

__device__ void TileRenderer::drawFilled(const uint32_t tile)
{
    static_assert(LIBFIVE_CUDA_TILE_SIZE_PX >= 16, "Tiles are too small");
    static_assert(LIBFIVE_CUDA_TILE_SIZE_PX % 16 == 0, "Invalid tile size");

    // Convert from tile position to pixels
    const uint32_t px = (tile / tiles.per_side) * LIBFIVE_CUDA_TILE_SIZE_PX;
    const uint32_t py = (tile % tiles.per_side) * LIBFIVE_CUDA_TILE_SIZE_PX;

    uint4* pix = reinterpret_cast<uint4*>(&image[px + py * image.size_px]);
    const uint4 fill = make_uint4(0xB0B0B0B0, 0xB0B0B0B0, 0xB0B0B0B0, 0xB0B0B0B0);
    for (unsigned y=0; y < LIBFIVE_CUDA_TILE_SIZE_PX; y++) {
        for (unsigned x=0; x < LIBFIVE_CUDA_TILE_SIZE_PX; x += 16) {
            *pix = fill;
            pix++;
        }
        pix += (image.size_px - LIBFIVE_CUDA_TILE_SIZE_PX) / 16;
    }
}

__global__ void TileRenderer_drawFilled(TileRenderer* r, const uint32_t offset)
{
    // Each thread picks a block and fills in the whole thing
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t start = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t i = start + offset;
    if (i < r->tiles.num_filled) {
        const uint32_t tile = r->tiles.filled(i);
        r->drawFilled(tile);
    }
}

////////////////////////////////////////////////////////////////////////////////

SubtileRenderer::SubtileRenderer(const Tape& tape, Image& image,
                                 TileRenderer& prev)
    : tape(tape), image(image), tiles(prev.tiles),
      subtiles(image.size_px, LIBFIVE_CUDA_SUBTILE_SIZE_PX),

      regs_lower(CUDA_MALLOC(Registers,
        LIBFIVE_CUDA_SUBTILE_BLOCKS * tape.num_regs * 2)),
      regs_upper(regs_lower + LIBFIVE_CUDA_SUBTILE_BLOCKS * tape.num_regs),

      active(CUDA_MALLOC(ActiveArray,
                  LIBFIVE_CUDA_SUBTILE_BLOCKS * tape.num_regs)),
      choices(tape.num_csg_choices ?
              CUDA_MALLOC(ChoiceArray,
                  LIBFIVE_CUDA_SUBTILE_BLOCKS * tape.num_csg_choices)
              : nullptr)
{
    // Nothing to do here
}

SubtileRenderer::~SubtileRenderer()
{
    CHECK(hipFree(regs_lower));
    CHECK(hipFree(active));
    CHECK(hipFree(choices));
}

__device__
void SubtileRenderer::check(const uint32_t subtile,
                            const uint32_t tile,
                            const View& v)
{
    auto regs_lower = this->regs_lower + tape.num_regs * blockIdx.x;
    auto regs_upper = this->regs_upper + tape.num_regs * blockIdx.x;
    storeAxes(threadIdx.x, subtile, v, subtiles, tape, regs_lower, regs_upper);

    auto choices = this->choices + tape.num_csg_choices * blockIdx.x;

    // Run actual evaluation
    uint32_t subtape_index = tiles.head(tile);
    uint32_t s = tiles.subtapes.start[subtape_index];
    const Clause* __restrict__ tape = tiles.subtapes.data[subtape_index];
    const float* __restrict__ constant_ptr = &this->tape.constant(0);

    uint32_t next = tiles.subtapes.next[subtape_index];
    uint32_t next_start = tiles.subtapes.start[next];
    uint32_t length = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;

    // We copy LIBFIVE_CUDA_SUBTILES_PER_TILE clauses from each active tape
    // into shared memory, to speed up the first pass a little bit.  Beyond
    // that point, tapes diverge in size, so we can't realiably sync threads.
    __shared__ Clause local[LIBFIVE_CUDA_REFINE_TILES]
                           [LIBFIVE_CUDA_SUBTILES_PER_TILE];
    const auto u = threadIdx.x / LIBFIVE_CUDA_SUBTILES_PER_TILE;
    const uint32_t q = threadIdx.x % LIBFIVE_CUDA_SUBTILES_PER_TILE;
    if (s + q < LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE) {
        local[u][q] = tape[s + q];
    }

    {   // If this chunk is larger than the short cached tape, then
        // we'll set the next chunk to re-enter this chunk at a
        // later point to finish it up.
        const auto chunk_length = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE - s;
        if (chunk_length > LIBFIVE_CUDA_SUBTILES_PER_TILE) {
            length = LIBFIVE_CUDA_SUBTILES_PER_TILE;
            next = subtape_index;
            next_start = s + length;
        } else {
            // Otherwise, we'll finish the entire cached subtape
            length = chunk_length;
        }
    }

    // Reassign the first tape to our chunk of shared memory
    tape = local[u];
    s = 0;
    __syncthreads();

    Interval result;
    while (true) {
        using namespace libfive::Opcode;

        if (s == length) {
            if (next) {
                subtape_index = next;
                s = next_start;
                tape = tiles.subtapes.data[subtape_index];

                // Preload these values
                next = tiles.subtapes.next[subtape_index];
                next_start = tiles.subtapes.start[next];
                length = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
            } else {
                result.lower = regs_lower[tape[s - 1].out][threadIdx.x];
                result.upper = regs_upper[tape[s - 1].out][threadIdx.x];
                break;
            }
        }
        const Clause c = tape[s++];

        // All clauses must have at least one argument, since constants
        // and VAR_X/Y/Z are handled separately.
        Interval lhs;
        if (c.banks & 1) {
            const float f = constant_ptr[c.lhs];
            lhs.lower = f;
            lhs.upper = f;
        } else {
            lhs.lower = regs_lower[c.lhs][threadIdx.x];
            lhs.upper = regs_upper[c.lhs][threadIdx.x];
        }

        Interval rhs;
        if (c.banks & 2) {
            const float f = constant_ptr[c.rhs];
            rhs.lower = f;
            rhs.upper = f;
        } else if (c.opcode >= OP_ADD) {
            rhs.lower = regs_lower[c.rhs][threadIdx.x];
            rhs.upper = regs_upper[c.rhs][threadIdx.x];
        }

        Interval out = intervalOp(c.opcode, lhs, rhs, choices);
        regs_lower[c.out][threadIdx.x] = out.lower;
        regs_upper[c.out][threadIdx.x] = out.upper;
    }

    ////////////////////////////////////////////////////////////////////////////

    // Reverse the tape
    if ((threadIdx.x % LIBFIVE_CUDA_SUBTILES_PER_TILE) == 0) {
        uint32_t subtape_index = tiles.head(tile);
        uint32_t prev = 0;

        while (true) {
            const uint32_t next = tiles.subtapes.next[subtape_index];
            tiles.subtapes.next[subtape_index] = prev;
            if (next == 0) {
                break;
            } else {
                prev = subtape_index;
                subtape_index = next;
            }
        }
        tiles.head(tile) = subtape_index;
    }
    __syncthreads();

    // If this tile is unambiguously filled, then mark it at the end
    // of the tiles list
    if (result.upper < 0.0f) {
        subtiles.insert_filled(subtile);
        return;
    }

    // If the tile is ambiguous, then record it as needing further refinement
    else if ((result.lower <= 0.0f && result.upper >= 0.0f)
            || isnan(result.lower)
            || isnan(result.upper))
    {
        subtiles.insert_active(subtile);
    }

    else {
        return;
    }

    ////////////////////////////////////////////////////////////////////////////

    // Pick a subset of the active array to use for this block
    auto active = this->active + blockIdx.x * this->tape.num_regs;

    for (uint32_t r=0; r < this->tape.num_regs; ++r) {
        active[r][threadIdx.x] = false;
    }

    // The tape chunks must be reversed by this point!
    uint32_t in_subtape_index = tiles.head(tile);
    uint32_t in_s = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
    uint32_t in_s_end = tiles.subtapes.start[in_subtape_index];
    const Clause* __restrict__ in_tape = tiles.subtapes.data[in_subtape_index];

    // Mark the head of the tape as active
    active[in_tape[in_s - 1].out][threadIdx.x] = true;

    // Claim a subtape to populate
    uint32_t out_subtape_index = atomicAdd(&subtiles.num_subtapes, 1);
    assert(out_subtape_index < LIBFIVE_CUDA_NUM_SUBTAPES);
    uint32_t out_s = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
    Clause* __restrict__ out_tape = subtiles.subtapes.data[out_subtape_index];

    // Since we're reversing the tape, this is going to be the
    // end of the linked list (i.e. next = 0)
    subtiles.subtapes.next[out_subtape_index] = 0;

    while (true) {
        using namespace libfive::Opcode;

        // If we've reached the end of an input tape chunk, then
        // either move on to the next one or escape the loop
        if (in_s == in_s_end) {
            const uint32_t next = tiles.subtapes.next[in_subtape_index];
            if (next) {
                in_subtape_index = next;
                in_s = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
                in_s_end = tiles.subtapes.start[in_subtape_index];
                in_tape = tiles.subtapes.data[in_subtape_index];
            } else {
                break;
            }
        }
        Clause c = in_tape[--in_s];

        if (active[c.out][threadIdx.x]) {
            active[c.out][threadIdx.x] = false;
            if (c.opcode == OP_MIN || c.opcode == OP_MAX) {
                const uint8_t choice = (*(--choices))[threadIdx.x];
                if (choice == 1) {
                    if (!(c.banks & 1)) {
                        active[c.lhs][threadIdx.x] = true;
                        if (c.lhs == c.out) {
                            continue;
                        }
                        c.rhs = c.lhs;
                        c.banks = 0;
                    } else {
                        c.rhs = c.lhs;
                        c.banks = 3;
                    }
                } else if (choice == 2) {
                    if (!(c.banks & 2)) {
                        active[c.rhs][threadIdx.x] = true;
                        if (c.rhs == c.out) {
                            continue;
                        }
                        c.lhs = c.rhs;
                        c.banks = 0;
                    } else {
                        c.lhs = c.rhs;
                        c.banks = 3;
                    }
                } else if (choice == 0) {
                    if (!(c.banks & 1)) {
                        active[c.lhs][threadIdx.x] = true;
                    }
                    if (!(c.banks & 2)) {
                        active[c.rhs][threadIdx.x] = true;
                    }
                } else {
                    printf("Bad choice %u\n", choice);
                    assert(false);
                }
            } else {
                if (!(c.banks & 1)) {
                    active[c.lhs][threadIdx.x] = true;
                }
                if (c.opcode >= OP_ADD && !(c.banks & 2)) {
                    active[c.rhs][threadIdx.x] = true;
                }
            }

            // If we've reached the end of the output tape, then
            // allocate a new one and keep going
            if (out_s == 0) {
                const auto next = atomicAdd(&subtiles.num_subtapes, 1);
                subtiles.subtapes.start[out_subtape_index] = 0;
                subtiles.subtapes.next[next] = out_subtape_index;

                out_subtape_index = next;
                out_s = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
                out_tape = subtiles.subtapes.data[out_subtape_index];
            }

            out_tape[--out_s] = c;
        } else if (c.opcode == OP_MIN || c.opcode == OP_MAX) {
            --choices;
        }
    }

    // The last subtape may not be completely filled, so write its size here
    subtiles.subtapes.start[out_subtape_index] = out_s;
    subtiles.head(subtile) = out_subtape_index;
}

__global__
void SubtileRenderer_check(SubtileRenderer* r,
                           const uint32_t offset,
                           View v)
{
    assert(blockDim.x % LIBFIVE_CUDA_SUBTILES_PER_TILE == 0);
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    // Pick an active tile from the list.  Each block executes multiple tiles!
    const uint32_t stride = blockDim.x / LIBFIVE_CUDA_SUBTILES_PER_TILE;
    const uint32_t sub = threadIdx.x / LIBFIVE_CUDA_SUBTILES_PER_TILE;
    const uint32_t i = offset + blockIdx.x * stride + sub;

    if (i < r->tiles.num_active) {
        // Pick out the next active tile
        // (this will be the same for every thread in a block)
        const uint32_t tile = r->tiles.active(i);

        // Convert from tile position to pixels
        const uint32_t px = (tile / r->tiles.per_side) *
                            LIBFIVE_CUDA_TILE_SIZE_PX;
        const uint32_t py = (tile % r->tiles.per_side) *
                            LIBFIVE_CUDA_TILE_SIZE_PX;

        // Then convert from pixels into subtiles
        const uint32_t p = threadIdx.x % LIBFIVE_CUDA_SUBTILES_PER_TILE;
        const uint32_t dx = p % LIBFIVE_CUDA_SUBTILES_PER_TILE_SIDE;
        const uint32_t dy = p / LIBFIVE_CUDA_SUBTILES_PER_TILE_SIDE;

        const uint32_t tx = px / LIBFIVE_CUDA_SUBTILE_SIZE_PX + dx;
        const uint32_t ty = py / LIBFIVE_CUDA_SUBTILE_SIZE_PX + dy;

        // Finally, unconvert back into a single index
        const uint32_t subtile = ty + tx * r->subtiles.per_side;

        r->check(subtile, tile, v);
    }
}

__device__ void SubtileRenderer::drawFilled(const uint32_t tile)
{
    static_assert(LIBFIVE_CUDA_TILE_SIZE_PX >= 8, "Tiles are too small");
    static_assert(LIBFIVE_CUDA_TILE_SIZE_PX % 8 == 0, "Invalid tile size");

    // Convert from tile position to pixels
    const uint32_t px = (tile / subtiles.per_side) * LIBFIVE_CUDA_SUBTILE_SIZE_PX;
    const uint32_t py = (tile % subtiles.per_side) * LIBFIVE_CUDA_SUBTILE_SIZE_PX;

    uint2* pix = reinterpret_cast<uint2*>(&image[px + py * image.size_px]);
    const uint2 fill = make_uint2(0xD0D0D0D0, 0xD0D0D0D0);
    for (unsigned y=0; y < LIBFIVE_CUDA_SUBTILE_SIZE_PX; y++) {
        for (unsigned x=0; x < LIBFIVE_CUDA_SUBTILE_SIZE_PX; x += 8) {
            *pix = fill;
            pix++;
        }
        pix += (image.size_px - LIBFIVE_CUDA_SUBTILE_SIZE_PX) / 8;
    }
}

__global__ void SubtileRenderer_drawFilled(SubtileRenderer* r, const uint32_t offset)
{
    // Each thread picks a block and fills in the whole thing
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t start = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t i = start + offset;
    if (i < r->subtiles.num_filled) {
        const uint32_t tile = r->subtiles.filled(i);
        r->drawFilled(tile);
    }
}

////////////////////////////////////////////////////////////////////////////////

PixelRenderer::PixelRenderer(const Tape& tape, Image& image,
                             const SubtileRenderer& prev)
    : tape(tape), image(image), subtiles(prev.subtiles),
      regs(CUDA_MALLOC(FloatRegisters,
                       tape.num_regs * LIBFIVE_CUDA_RENDER_BLOCKS))
{
    // Nothing to do here
}

PixelRenderer::~PixelRenderer()
{
    CHECK(hipFree(regs));
}

__device__ void PixelRenderer::draw(const uint32_t subtile, const View& v)
{
    const uint32_t pixel = threadIdx.x % LIBFIVE_CUDA_PIXELS_PER_SUBTILE;
    const uint32_t dx = pixel % LIBFIVE_CUDA_SUBTILE_SIZE_PX;
    const uint32_t dy = pixel / LIBFIVE_CUDA_SUBTILE_SIZE_PX;

    // Pick an index into the register array
    auto regs = this->regs + tape.num_regs * blockIdx.x;

    // Convert from tile position to pixels
    uint32_t px = (subtile / subtiles.per_side) *
                   LIBFIVE_CUDA_SUBTILE_SIZE_PX + dx;
    uint32_t py = (subtile % subtiles.per_side) *
                   LIBFIVE_CUDA_SUBTILE_SIZE_PX + dy;

    {   // Prepopulate axis values
        const float x = px / (image.size_px - 1.0f);
        const float y = py / (image.size_px - 1.0f);
        float vs[3];
        vs[0] = 2.0f * (x - 0.5f) * v.scale - v.center[0];
        vs[1] = 2.0f * (y - 0.5f) * v.scale - v.center[1];
        vs[2] = 0.0f;
        for (unsigned i=0; i < 3; ++i) {
            if (tape.axes.reg[i] != UINT16_MAX) {
                regs[tape.axes.reg[i]][threadIdx.x] = vs[i];
            }
        }
    }

    uint32_t subtape_index = subtiles.head(subtile);
    uint32_t s = subtiles.subtapes.start[subtape_index];
    const float* __restrict__ constant_ptr = &tape.constant(0);
    const Clause* __restrict__ tape = subtiles.subtapes.data[subtape_index];

    while (true) {
        using namespace libfive::Opcode;

        // Move to the next subtape if this one is finished
        if (s == LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE) {
            const uint32_t next = subtiles.subtapes.next[subtape_index];
            if (next) {
                subtape_index = next;
                s = subtiles.subtapes.start[subtape_index];
                tape = subtiles.subtapes.data[subtape_index];
            } else {
                if (regs[tape[s - 1].out][threadIdx.x] < 0.0f) {
                    image(px, py) = 255;
                }
                return;
            }
        }
        const Clause c = tape[s++];

        // All clauses must have at least one argument, since constants
        // and VAR_X/Y/Z are handled separately.
        float lhs;
        if (c.banks & 1) {
            lhs = constant_ptr[c.lhs];
        } else {
            lhs = regs[c.lhs][threadIdx.x];
        }

        float rhs;
        if (c.banks & 2) {
            rhs = constant_ptr[c.rhs];
        } else if (c.opcode >= OP_ADD) {
            rhs = regs[c.rhs][threadIdx.x];
        }

        float out;
        switch (c.opcode) {
            case OP_SQUARE: out = lhs * lhs; break;
            case OP_SQRT: out = sqrtf(lhs); break;
            case OP_NEG: out = -lhs; break;
            // Skipping transcendental functions for now

            case OP_ADD: out = lhs + rhs; break;
            case OP_MUL: out = lhs * rhs; break;
            case OP_DIV: out = lhs / rhs; break;
            case OP_MIN: out = fminf(lhs, rhs); break;
            case OP_MAX: out = fmaxf(lhs, rhs); break;
            case OP_SUB: out = lhs - rhs; break;

            // Skipping various hard functions here
            default: break;
        }
        regs[c.out][threadIdx.x] = out;
    }
}

__global__ void PixelRenderer_draw(PixelRenderer* r,
                                   const Tiles* subtiles,
                                   const uint32_t offset, View v)
{
    // We assume one thread per pixel in a tile
    assert(blockDim.x % LIBFIVE_CUDA_SUBTILE_SIZE_PX == 0);
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    // Pick an active tile from the list.  Each block executes multiple tiles!
    const uint32_t stride = blockDim.x / LIBFIVE_CUDA_PIXELS_PER_SUBTILE;
    const uint32_t sub = threadIdx.x / LIBFIVE_CUDA_PIXELS_PER_SUBTILE;
    const uint32_t i = offset + blockIdx.x * stride + sub;

    if (i < subtiles->num_active) {
        const uint32_t subtile = subtiles->active(i);
        r->draw(subtile, v);
    }
}

////////////////////////////////////////////////////////////////////////////////
__global__
void Renderable_copyToTexture(Renderable* r, bool append, hipSurfaceObject_t surf)
{
    unsigned x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned y = threadIdx.y + blockIdx.y * blockDim.y;

    const unsigned size = r->image.size_px;
    if (x < size && y < size) {
        const uint8_t c = r->image(x, size - y - 1);
        if (c) {
            surf2Dwrite(0x00FFFFFF | (c << 24), surf, x*4, y);
        } else if (!append) {
            surf2Dwrite(0, surf, x*4, y);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////

void Renderable::Deleter::operator()(Renderable* r)
{
    r->~Renderable();
    CHECK(hipFree(r));
}

Renderable::~Renderable()
{
    for (auto& s : streams) {
        CHECK(hipStreamDestroy(s));
    }
}

Renderable::Handle Renderable::build(libfive::Tree tree, uint32_t image_size_px)
{
    auto out = CUDA_MALLOC(Renderable, 1);
    new (out) Renderable(tree, image_size_px);
    hipDeviceSynchronize();
    return Handle(out);
}

Renderable::Renderable(libfive::Tree tree, uint32_t image_size_px)
    : image(image_size_px),
      tape(std::move(Tape::build(tree))),

      tile_renderer(tape, image),
      subtile_renderer(tape, image, tile_renderer),
      pixel_renderer(tape, image, subtile_renderer)
{
    CHECK(hipStreamCreate(&streams[0]));
    CHECK(hipStreamCreate(&streams[1]));
}

void Renderable::run(const View& view)
{
    hipStream_t streams[2] = {this->streams[0], this->streams[1]};

    // Record this local variable because otherwise it looks up memory
    // that has been loaned to the GPU and not synchronized.
    TileRenderer* tile_renderer = &this->tile_renderer;
    const uint32_t total_tiles = tile_renderer->tiles.total;
    const uint32_t tile_stride = LIBFIVE_CUDA_TILE_THREADS *
                                 LIBFIVE_CUDA_TILE_BLOCKS;
    SubtileRenderer* subtile_renderer = &this->subtile_renderer;
    PixelRenderer* pixel_renderer = &this->pixel_renderer;
    auto tiles = &tile_renderer->tiles;
    auto subtiles = &subtile_renderer->subtiles;

    // Reset everything in preparation for a render
    tiles->reset();
    subtiles->reset();
    hipMemset(image.data, 0, image.size_px * image.size_px);

    // Do per-tile evaluation to get filled / ambiguous tiles
    for (unsigned i=0; i < total_tiles; i += tile_stride) {
        TileRenderer_check<<<LIBFIVE_CUDA_TILE_BLOCKS,
                             LIBFIVE_CUDA_TILE_THREADS,
                             0, streams[0]>>>(tile_renderer, i, view);
    }
    hipDeviceSynchronize();

    // Pull a few variables back from the GPU
    const uint32_t filled_tiles = tiles->num_filled;
    const uint32_t active_tiles = tiles->num_active;

    for (unsigned i=0; i < filled_tiles; i += tile_stride) {
        // Drawing filled and ambiguous tiles can happen simultaneously,
        // so we assign each one to a separate stream.
        TileRenderer_drawFilled<<<LIBFIVE_CUDA_TILE_BLOCKS,
                                  LIBFIVE_CUDA_TILE_THREADS,
                                  0, streams[1]>>>(tile_renderer, i);
    }

    // Refine ambiguous tiles from their subtapes
    const uint32_t subtile_check_stride = LIBFIVE_CUDA_SUBTILE_BLOCKS *
                                          LIBFIVE_CUDA_REFINE_TILES;
    for (unsigned i=0; i < active_tiles; i += subtile_check_stride) {
        SubtileRenderer_check<<<LIBFIVE_CUDA_SUBTILE_BLOCKS,
            LIBFIVE_CUDA_SUBTILES_PER_TILE *
            LIBFIVE_CUDA_REFINE_TILES,
            0, streams[0]>>>(
                    subtile_renderer, i, view);
    }

    hipDeviceSynchronize();

    const uint32_t filled_subtiles = subtile_renderer->subtiles.num_filled;
    const uint32_t active_subtiles = subtile_renderer->subtiles.num_active;
    const uint32_t subtile_stride = LIBFIVE_CUDA_SUBTILE_BLOCKS *
                                    LIBFIVE_CUDA_SUBTILE_THREADS;
    for (unsigned i=0; i < filled_subtiles; i += subtile_stride) {
        SubtileRenderer_drawFilled<<<LIBFIVE_CUDA_SUBTILE_BLOCKS,
                                     LIBFIVE_CUDA_SUBTILE_THREADS,
                                     0, streams[1]>>>(
            subtile_renderer, i);
    }

    // Do pixel-by-pixel rendering for active subtiles
    const uint32_t subtile_render_stride = LIBFIVE_CUDA_RENDER_BLOCKS *
                                           LIBFIVE_CUDA_RENDER_SUBTILES;
    for (unsigned i=0; i < active_subtiles; i += subtile_render_stride) {
        PixelRenderer_draw<<<LIBFIVE_CUDA_RENDER_BLOCKS,
                             LIBFIVE_CUDA_PIXELS_PER_SUBTILE *
                             LIBFIVE_CUDA_RENDER_SUBTILES, 0, streams[0]>>>(
            pixel_renderer, subtiles, i, view);
    }
    hipDeviceSynchronize();
}

hipGraphicsResource* Renderable::registerTexture(GLuint t)
{
    hipGraphicsResource* gl_tex;
    CHECK(hipGraphicsGLRegisterImage(&gl_tex, t, GL_TEXTURE_2D,
                                      cudaGraphicsMapFlagsWriteDiscard));
    return gl_tex;
}

void Renderable::copyToTexture(hipGraphicsResource* gl_tex, bool append)
{
    hipArray* array;
    CHECK(hipGraphicsMapResources(1, &gl_tex));
    CHECK(hipGraphicsSubResourceGetMappedArray(&array, gl_tex, 0, 0));

    // Specify texture
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = array;

    // Surface object??!
    hipSurfaceObject_t surf = 0;
    CHECK(hipCreateSurfaceObject(&surf, &res_desc));

    CHECK(hipDeviceSynchronize());
    Renderable_copyToTexture<<<dim3(256, 256), dim3(16, 16)>>>(
            this, append, surf);
    CHECK(hipGetLastError());

    CHECK(hipDeviceSynchronize());
    CHECK(hipDestroySurfaceObject(surf));
    CHECK(hipGraphicsUnmapResources(1, &gl_tex));
}
