#include "hip/hip_runtime.h"
#include "check.hpp"
#include "renderable.hpp"
#include "gpu_interval.hpp"
#include "parameters.hpp"

__constant__ static uint64_t const_buffer[0x2000];

void Renderable::Deleter::operator()(Renderable* r)
{
    r->~Renderable();
    CHECK(hipFree(r));
}

Renderable::~Renderable()
{
    CHECK(hipFree(scratch));
    CHECK(hipFree(tiles));
    CHECK(hipFree(subtapes));
    CHECK(hipFree(image));
    for (auto& s : streams) {
        CHECK(hipStreamDestroy(s));
    }
}

Renderable::Handle Renderable::build(libfive::Tree tree, uint32_t image_size_px)
{
    auto out = CUDA_MALLOC(Renderable, 1);
    new (out) Renderable(tree, image_size_px);
    return std::unique_ptr<Renderable, Deleter>(out);
}

Renderable::Renderable(libfive::Tree tree, uint32_t image_size_px)
    : tape(std::move(Tape::build(tree))),

      IMAGE_SIZE_PX(image_size_px),
      TILE_COUNT(IMAGE_SIZE_PX / LIBFIVE_CUDA_TILE_SIZE_PX),
      TOTAL_TILES(TILE_COUNT * TILE_COUNT),

      scratch(CUDA_MALLOC(uint8_t,
          std::max(LIBFIVE_CUDA_TILE_BLOCKS * LIBFIVE_CUDA_TILE_THREADS *
                           sizeof(Interval) * tape.num_regs
                       + TOTAL_TILES * max(1, tape.num_csg_choices),
                   sizeof(float) * tape.num_regs * LIBFIVE_CUDA_RENDER_BLOCKS
                                 * LIBFIVE_CUDA_TILE_SIZE_PX
                                 * LIBFIVE_CUDA_TILE_SIZE_PX))),
      regs_i(reinterpret_cast<IntervalRegisters*>(scratch)),
      csg_choices(scratch + LIBFIVE_CUDA_TILE_BLOCKS * LIBFIVE_CUDA_TILE_THREADS
                            * sizeof(Interval) * tape.num_regs),
      regs_f(reinterpret_cast<float*>(scratch)),

      tiles(CUDA_MALLOC(uint32_t, 2 * TOTAL_TILES)),
      active_tiles(0),
      filled_tiles(0),

      subtapes(CUDA_MALLOC(Subtape, LIBFIVE_CUDA_NUM_SUBTAPES)),
      active_subtapes(1),

      image(CUDA_MALLOC(uint8_t, IMAGE_SIZE_PX * IMAGE_SIZE_PX))
{
    hipMemset(image, 0, IMAGE_SIZE_PX * IMAGE_SIZE_PX);
    CHECK(hipStreamCreate(&streams[0]));
    CHECK(hipStreamCreate(&streams[1]));
}

////////////////////////////////////////////////////////////////////////////////

__device__ Interval walkI(const Tape& tape,
                          Renderable::IntervalRegisters* const __restrict__ fast_regs,
                          Renderable::IntervalRegisters* const __restrict__ regs,
                          uint8_t* const __restrict__ choices)
{
    using namespace libfive::Opcode;

    uint32_t choice_index = 0;

    const Clause* __restrict__ clause_ptr = &tape[0];
    const float* __restrict__ constant_ptr = &tape.constant(0);
    const uint32_t num_clauses = tape.num_clauses;

    for (uint32_t i=0; i < num_clauses; ++i) {
        const Clause c = clause_ptr[i];
        // All clauses must have at least one argument, since constants
        // and VAR_X/Y/Z are handled separately.
        Interval lhs;
        if (c.banks & 1) {
            const float f = constant_ptr[c.lhs];
            lhs.lower = f;
            lhs.upper = f;
        } else if (c.lhs < LIBFIVE_CUDA_FAST_REG_COUNT) {
            lhs = fast_regs[c.lhs][threadIdx.x];
        } else {
            lhs = regs[c.lhs][threadIdx.x];
        }

        Interval rhs;
        if (c.opcode >= OP_ADD) {
            if (c.banks & 2) {
                const float f = constant_ptr[c.rhs];
                rhs.lower = f;
                rhs.upper = f;
            } else if (c.rhs < LIBFIVE_CUDA_FAST_REG_COUNT) {
                rhs = fast_regs[c.rhs][threadIdx.x];
            } else {
                rhs = regs[c.rhs][threadIdx.x];
            }
        }

        Interval out;
        switch (c.opcode) {
            case OP_SQUARE: out = lhs.square(); break;
            case OP_SQRT: out = lhs.sqrt(); break;
            case OP_NEG: out = -lhs; break;
            // Skipping transcendental functions for now

            case OP_ADD: out = lhs + rhs; break;
            case OP_MUL: out = lhs * rhs; break;
            case OP_DIV: out = lhs / rhs; break;
            case OP_MIN: if (lhs.upper < rhs.lower) {
                             choices[choice_index] = 1;
                             out = lhs;
                         } else if (rhs.upper < lhs.lower) {
                             choices[choice_index] = 2;
                             out = rhs;
                         } else {
                             choices[choice_index] = 0;
                             out = lhs.min(rhs);
                         }
                         choice_index++;
                         break;
            case OP_MAX: if (lhs.lower > rhs.upper) {
                             choices[choice_index] = 1;
                             out = lhs;
                         } else if (rhs.lower > lhs.upper) {
                             choices[choice_index] = 2;
                             out = rhs;
                         } else {
                             choices[choice_index] = 0;
                             out = lhs.max(rhs);
                         }
                         choice_index++;
                         break;
            case OP_SUB: out = lhs - rhs; break;

            // Skipping various hard functions here
            default: break;
        }

        if (c.out < LIBFIVE_CUDA_FAST_REG_COUNT) {
            fast_regs[c.out][threadIdx.x] = out;
        } else {
            regs[c.out][threadIdx.x] = out;
        }
    }
    // Copy output to standard register before exiting
    const Clause c = clause_ptr[num_clauses - 1];
    if (c.out < LIBFIVE_CUDA_FAST_REG_COUNT) {
        return fast_regs[c.out][threadIdx.x];
    } else {
        return regs[c.out][threadIdx.x];
    }
}

__device__
void Renderable::processTiles(const uint32_t offset, const View& v)
{
    // This should be a 1D kernel
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t start = threadIdx.x + blockIdx.x * blockDim.x;

    const uint32_t index = start + offset;
    if (index >= TOTAL_TILES) {
        return;
    }
    __shared__ IntervalRegisters fast_regs[LIBFIVE_CUDA_FAST_REG_COUNT];

    {   // Prepopulate axis values
        const float x = index / TILE_COUNT;
        const float y = index % TILE_COUNT;

        Interval vs[3];
        const Interval X = {x / TILE_COUNT, (x + 1) / TILE_COUNT};
        vs[0].lower = 2.0f * (X.lower - 0.5f - v.center[0]) * v.scale;
        vs[0].upper = 2.0f * (X.upper - 0.5f - v.center[0]) * v.scale;

        const Interval Y = {y / TILE_COUNT, (y + 1) / TILE_COUNT};
        vs[1].lower = 2.0f * (Y.lower - 0.5f - v.center[1]) * v.scale;
        vs[1].upper = 2.0f * (Y.upper - 0.5f - v.center[1]) * v.scale;

        vs[2].lower = 0.0f;
        vs[2].upper = 0.0f;

        for (unsigned i=0; i < 3; ++i) {
            if (tape.axes.reg[i] != UINT16_MAX) {
                if (tape.axes.reg[i] < LIBFIVE_CUDA_FAST_REG_COUNT) {
                    fast_regs[tape.axes.reg[i]][threadIdx.x] = vs[i];
                } else {
                    regs_i[tape.axes.reg[i]][threadIdx.x] = vs[i];
                }
            }
        }
    }

    // Unpack a 1D offset into the data arrays
    auto csg_choices = this->csg_choices + index * tape.num_csg_choices;

    // Run actual evaluation
    const Interval result = walkI(
            tape, fast_regs, regs_i + tape.num_regs * blockIdx.x, csg_choices);

    // If this tile is unambiguously filled, then mark it at the end
    // of the tiles list
    if (result.upper < 0.0f) {
        const uint32_t i = atomicAdd(&filled_tiles, 1);
        tiles[TOTAL_TILES*2 - 1 - i] = index;
    }

    // If the tile is ambiguous, then record it as needing further refinement
    else if (result.lower <= 0.0f && result.upper >= 0.0f) {
        // Store the linked list of subtapes into the active tiles list
        const uint32_t i = atomicAdd(&active_tiles, 1);
        tiles[2 * i] = index;
    }
}

__global__ void processTiles(Renderable* r, const uint32_t offset,
                             Renderable::View v)
{
    r->processTiles(offset, v);
}

__device__
void Renderable::buildSubtapes(const uint32_t offset)
{
    // This is a 1D kernel which consumes tiles and writes out their tapes
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t start = threadIdx.x + blockIdx.x * blockDim.x;

    // Reuse the registers array to track activeness
    const uint32_t i = start + offset;
    if (i >= active_tiles) {
        return;
    }
    const uint32_t index = tiles[2 * i];

    bool* __restrict__ active = reinterpret_cast<bool*>(regs_i) + start * tape.num_regs;
    for (uint32_t j=0; j < tape.num_regs; ++j) {
        active[j] = false;
    }

    // Pick an offset CSG choices array
    auto csg_choices = this->csg_choices + index * tape.num_csg_choices;

    // Mark the root of the tree as true
    uint32_t t = tape.num_clauses;
    active[tape[t - 1].out] = true;

    // Begin walking down CSG choices
    uint32_t csg_choice = tape.num_csg_choices;

    // Claim a subtape to populate
    uint32_t subtape_index = atomicAdd(&active_subtapes, 1);
    assert(subtape_index < LIBFIVE_CUDA_NUM_SUBTAPES);

    // Since we're reversing the tape, this is going to be the
    // end of the linked list (i.e. next = 0)
    Subtape* subtape = &subtapes[subtape_index];
    subtape->next = 0;
    uint32_t s = 0;

    // Walk from the root of the tape downwards
    while (t--) {
        using namespace libfive::Opcode;
        const Clause c = tape[t];
        if (active[c.out]) {
            active[c.out] = false;
            uint32_t mask = 0;
            if (c.opcode == OP_MIN || c.opcode == OP_MAX)
            {
                const uint8_t choice = csg_choices[--csg_choice];
                if (choice == 1) {
                    if (!(c.banks & 1)) {
                        active[c.lhs] = true;
                        if (c.lhs == c.out) {
                            continue;
                        }
                    }
                } else if (choice == 2) {
                    active[c.rhs] = true;
                    if (!(c.banks & 2)) {
                        if (c.rhs == c.out) {
                            continue;
                        }
                    }
                } else if (choice == 0) {
                    if (!(c.banks & 1)) {
                        active[c.lhs] = true;
                    }
                    if (!(c.banks & 2)) {
                        active[c.rhs] = true;
                    }
                } else {
                    assert(false);
                }
                mask = (choice << 30);
            } else if (c.opcode >= OP_ADD) {
                if (!(c.banks & 1)) {
                    active[c.lhs] = true;
                }
                if (!(c.banks & 2)) {
                    active[c.rhs] = true;
                }
            } else if (c.opcode >= OP_SQUARE) {
                if (!(c.banks & 1)) {
                    active[c.lhs] = true;
                }
            }

            if (s == LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE) {
                auto next_subtape_index = atomicAdd(&active_subtapes, 1);
                auto next_subtape = &subtapes[next_subtape_index];
                subtape->size = LIBFIVE_CUDA_SUBTAPE_CHUNK_SIZE;
                next_subtape->next = subtape_index;

                subtape_index = next_subtape_index;
                subtape = next_subtape;
                s = 0;
            }
            (*subtape)[s++] = (t | mask);
        } else if (c.opcode == OP_MIN || c.opcode == OP_MAX) {
            --csg_choice;
        }
    }
    // The last subtape may not be completely filled
    subtape->size = s;

    // Store the linked list of subtapes into the active tiles list
    tiles[2 * i + 1] = subtape_index;
}

__global__ void buildSubtapes(Renderable* r, const uint32_t offset) {
    r->buildSubtapes(offset);
}

////////////////////////////////////////////////////////////////////////////////

__global__ void drawFilledTiles(Renderable* r, const uint32_t offset, Renderable::View v) {
    r->drawFilledTiles(offset, v);
}

__device__ void Renderable::drawFilledTiles(const uint32_t offset, const View& v)
{
    // Each thread picks a block and fills in the whole thing
    assert(blockDim.y == 1);
    assert(blockDim.z == 1);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t start = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t i = start + offset;
    if (i >= filled_tiles) {
        return;
    }

    const uint32_t tile = tiles[TOTAL_TILES*2 - i - 1];

    // Convert from tile position to pixels
    const uint32_t px = (tile / TILE_COUNT) * LIBFIVE_CUDA_TILE_SIZE_PX;
    const uint32_t py = (tile % TILE_COUNT) * LIBFIVE_CUDA_TILE_SIZE_PX;

    uint4* pix = reinterpret_cast<uint4*>(&image[px + py * IMAGE_SIZE_PX]);
    const uint4 fill = make_uint4(0xF0F0F0F0, 0xF0F0F0F0, 0xF0F0F0F0, 0xF0F0F0F0);
    for (unsigned y=0; y < LIBFIVE_CUDA_TILE_SIZE_PX; y++) {
        for (unsigned x=0; x < LIBFIVE_CUDA_TILE_SIZE_PX; x += 16) {
            *pix = fill;
            pix++;
        }
        pix += (IMAGE_SIZE_PX - LIBFIVE_CUDA_TILE_SIZE_PX) / 16;
    }
}

////////////////////////////////////////////////////////////////////////////////

__device__ float walkF(const Tape& tape,
                       const Subtape* const subtapes,
                       uint32_t subtape_index,
                       float* const __restrict__ regs)
{
    assert(subtape_index != 0);
    uint32_t s = subtapes[subtape_index].size;
    uint32_t target;
    while (true) {
        if (s == 0) {
            if (subtapes[subtape_index].next) {
                subtape_index = subtapes[subtape_index].next;
                s = subtapes[subtape_index].size;
            } else {
                return regs[tape[target].out];
            }
        }
        s -= 1;

        // Pick the target, which is an offset into the original tape
        target = subtapes[subtape_index][s];

        // Mask out choice bits
        const uint8_t choice = (target >> 30);
        target &= (1 << 30) - 1;

        const Clause c = tape[target];

#define LHS (!(c.banks & 1) ? regs[c.lhs] : tape.constant(c.lhs))
#define RHS (!(c.banks & 2) ? regs[c.rhs] : tape.constant(c.rhs))
        using namespace libfive::Opcode;
        switch (c.opcode) {
            case OP_SQUARE: regs[c.out] = LHS * LHS; break;
            case OP_SQRT: regs[c.out] = sqrtf(LHS); break;
            case OP_NEG: regs[c.out] = -LHS; break;
            // Skipping transcendental functions for now

            case OP_ADD: regs[c.out] = LHS + RHS; break;
            case OP_MUL: regs[c.out] = LHS * RHS; break;
            case OP_DIV: regs[c.out] = LHS / RHS; break;
            case OP_MIN: if (choice == 1) {
                            regs[c.out] = LHS;
                        } else if (choice == 2) {
                            regs[c.out] = RHS;
                        } else {
                            regs[c.out] = fminf(LHS, RHS);
                        }
                        break;
            case OP_MAX: if (choice == 1) {
                           regs[c.out] = LHS;
                        } else if (choice == 2) {
                           regs[c.out] = RHS;
                        } else {
                           regs[c.out] = fmaxf(LHS, RHS);
                        }
                        break;
            case OP_SUB: regs[c.out] = LHS - RHS; break;

            // Skipping various hard functions here
            default: break;
        }
    }
#undef LHS
#undef RHS
    assert(false);
    return 0.0f;
}

__device__ void Renderable::drawAmbiguousTiles(const uint32_t offset, const View& v)
{
    // We assume one thread per pixel in a tile
    assert(blockDim.x == LIBFIVE_CUDA_TILE_SIZE_PX);
    assert(blockDim.x == LIBFIVE_CUDA_TILE_SIZE_PX);
    assert(gridDim.y == 1);
    assert(gridDim.z == 1);

    const uint32_t dx = threadIdx.x;
    const uint32_t dy = threadIdx.y;

    // Pick an index into the register array
    const uint32_t pos = (blockIdx.x * LIBFIVE_CUDA_TILE_SIZE_PX + dx) *
                          LIBFIVE_CUDA_TILE_SIZE_PX + dy;
    float* const __restrict__ regs = regs_f + pos * tape.num_regs;

    // Pick an active tile from the list
    const uint32_t i = offset + blockIdx.x;
    if (i >= active_tiles) {
        return;
    }
    const uint32_t tile = tiles[i * 2];
    const uint32_t subtape_index = tiles[i * 2 + 1];

    // Convert from tile position to pixels
    uint32_t px = (tile / TILE_COUNT) * LIBFIVE_CUDA_TILE_SIZE_PX + dx;
    uint32_t py = (tile % TILE_COUNT) * LIBFIVE_CUDA_TILE_SIZE_PX + dy;

    {   // Prepopulate axis values
        const float x = px / (IMAGE_SIZE_PX - 1.0f);
        const float y = py / (IMAGE_SIZE_PX - 1.0f);
        float vs[3];
        vs[0] = 2.0f * (x - 0.5f - v.center[0]) * v.scale;
        vs[1] = 2.0f * (y - 0.5f - v.center[1]) * v.scale;
        vs[2] = 0.0f;
        for (unsigned i=0; i < 3; ++i) {
            if (tape.axes.reg[i] != UINT16_MAX) {
                regs[tape.axes.reg[i]] = vs[i];
            }
        }
    }
    const float f = walkF(tape, subtapes, subtape_index, regs);
    if (f < 0.0f) {
        image[px + py * IMAGE_SIZE_PX] = 255;
    }
}

__global__ void drawAmbiguousTiles(Renderable* r, const uint32_t offset,
                                   Renderable::View v)
{
    r->drawAmbiguousTiles(offset, v);
}

////////////////////////////////////////////////////////////////////////////////

void Renderable::run(const View& view)
{
    hipStream_t streams[2] = {this->streams[0], this->streams[1]};

    // Reset our counter variables
    active_tiles = 0;
    filled_tiles = 0;
    active_subtapes = 1;

    // Record this local variable because otherwise it looks up memory
    // that has been loaned to the GPU and not synchronized.
    const uint32_t total_tiles = TOTAL_TILES;
    const uint32_t stride = LIBFIVE_CUDA_TILE_THREADS *
                            LIBFIVE_CUDA_TILE_BLOCKS;

    tape.sendToConstantMemory((const char*)const_buffer);

    // Do per-tile evaluation to get filled / ambiguous tiles
    for (unsigned i=0; i < total_tiles; i += stride) {
        ::processTiles<<<LIBFIVE_CUDA_TILE_BLOCKS,
                         LIBFIVE_CUDA_TILE_THREADS,
                         0, streams[0]>>>(this, i, view);
        CHECK(hipGetLastError());
    }
    CHECK(hipStreamSynchronize(streams[0]));

    // Pull a few variables back from the GPU
    const uint32_t filled_tiles = this->filled_tiles;
    const uint32_t active_tiles = this->active_tiles;

    for (unsigned i=0; i < filled_tiles; i += stride) {
        // Drawing filled and ambiguous tiles can happen simultaneously,
        // so we assign each one to a separate stream.
        ::drawFilledTiles<<<LIBFIVE_CUDA_TILE_BLOCKS,
                            LIBFIVE_CUDA_TILE_THREADS,
                            0, streams[1]>>>(this, i, view);
        CHECK(hipGetLastError());
    }

    // Build subtapes in memory for ambiguous tiles
    for (unsigned i=0; i < active_tiles; i += stride) {
        ::buildSubtapes<<<LIBFIVE_CUDA_TILE_BLOCKS,
                          LIBFIVE_CUDA_TILE_THREADS,
                          0, streams[0]>>>(this, i);
        CHECK(hipGetLastError());
    }

    // Do pixel-by-pixel rendering for ambiguous tiles
    for (unsigned i=0; i < active_tiles; i += LIBFIVE_CUDA_RENDER_BLOCKS) {
        const dim3 T(LIBFIVE_CUDA_TILE_SIZE_PX, LIBFIVE_CUDA_TILE_SIZE_PX);
        ::drawAmbiguousTiles<<<LIBFIVE_CUDA_RENDER_BLOCKS,
                               T, 0, streams[0]>>>(this, i, view);
        CHECK(hipGetLastError());
    }
}
