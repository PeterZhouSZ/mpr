#include "hip/hip_runtime.h"
#include <cassert>

#include "libfive/tree/cache.hpp"

#include "v3.hpp"
#include "check.hpp"
#include "gpu_interval.hpp"
#include "gpu_opcode.hpp"

////////////////////////////////////////////////////////////////////////////////

// No need for parameters.hpp, we want to compile faster
// (without rebuilding everything else)
#define NUM_TILES (2)
#define NUM_THREADS (64 * NUM_TILES)
#define NUM_BLOCKS (512)
#define SUBTAPE_CHUNK_SIZE 64
#define NUM_SUBTAPES 320000

////////////////////////////////////////////////////////////////////////////////
// COPYPASTA
#define OP(d) (((uint8_t*)d)[0])
#define I_OUT(d) (((uint8_t*)d)[1])
#define I_LHS(d) (((uint8_t*)d)[2])
#define I_RHS(d) (((uint8_t*)d)[3])
#define IMM(d) (((float*)d)[1])
#define JUMP_TARGET(d) (((int32_t*)d)[1])

static __device__ void copy_imm_i(const uint64_t data,
                                  Interval* const __restrict__ slots)
{
    const float lhs = IMM(&data);
    const uint8_t i_out = I_OUT(&data);
    slots[i_out] = {lhs, lhs};
}

static __device__ void copy_imm_f(const uint64_t data,
                                  float* const __restrict__ slots)
{
    const float lhs = IMM(&data);
    const uint8_t i_out = I_OUT(&data);
    slots[i_out] = lhs;
}

static __device__ void copy_lhs_i(const uint64_t data,
                                  Interval* const __restrict__ slots)
{
    const uint8_t i_lhs = I_LHS(&data);
    const uint8_t i_out = I_OUT(&data);
    slots[i_out] = slots[i_lhs];
}

static __device__ void copy_lhs_f(const uint64_t data,
                                  float* const __restrict__ slots)
{
    const uint8_t i_lhs = I_LHS(&data);
    const uint8_t i_out = I_OUT(&data);
    slots[i_out] = slots[i_lhs];
}

static __device__ void copy_rhs_i(const uint64_t data,
                                  Interval* const __restrict__ slots)
{
    const uint8_t i_rhs = I_RHS(&data);
    const uint8_t i_out = I_OUT(&data);
    slots[i_out] = slots[i_rhs];
}

static __device__ void copy_rhs_f(const uint64_t data,
                                  float* const __restrict__ slots)
{
    const uint8_t i_rhs = I_RHS(&data);
    const uint8_t i_out = I_OUT(&data);
    slots[i_out] = slots[i_rhs];
}

#define FUNCTION_PREAMBLE_LHS(name, T, suffix)              \
static __device__                                           \
void name##_lhs_##suffix(const uint64_t data,               \
                    T* const __restrict__ slots)            \
{                                                           \
    const uint8_t i_lhs = I_LHS(&data);                     \
    const T lhs = slots[i_lhs];                             \
    const uint8_t i_out = I_OUT(&data);                     \

#define FUNCTION_PREAMBLE_IMM_RHS(name, T, suffix)          \
static __device__                                           \
void name##_imm_rhs_##suffix(const uint64_t data,           \
                    T* const __restrict__ slots)            \
{                                                           \
    const float lhs = IMM(&data);                           \
    const uint8_t i_rhs = I_RHS(&data);                     \
    const T rhs = slots[i_rhs];                             \
    const uint8_t i_out = I_OUT(&data);                     \

#define FUNCTION_PREAMBLE_LHS_IMM(name, T, suffix)          \
static __device__                                           \
void name##_lhs_imm_##suffix(const uint64_t data,           \
                    T* const __restrict__ slots)            \
{                                                           \
    const float rhs = IMM(&data);                           \
    const uint8_t i_lhs = I_LHS(&data);                     \
    const T lhs = slots[i_lhs];                             \
    const uint8_t i_out = I_OUT(&data);                     \

#define FUNCTION_PREAMBLE_LHS_RHS(name, T, suffix)          \
static __device__                                           \
void name##_lhs_rhs_##suffix(const uint64_t data,           \
                    T* const __restrict__ slots)            \
{                                                           \
    const uint8_t i_lhs = I_LHS(&data);                     \
    const T lhs = slots[i_lhs];                             \
    const uint8_t i_rhs = I_RHS(&data);                     \
    const T rhs = slots[i_rhs];                             \
    const uint8_t i_out = I_OUT(&data);                     \

// Special implementations of min and max, which manipulate the choices array
FUNCTION_PREAMBLE_LHS_IMM(min, float, f)
    slots[i_out] = fminf(lhs, rhs);
}
FUNCTION_PREAMBLE_LHS_IMM(min, Interval, i)
    uint8_t choice = 0;
    slots[i_out] = min(lhs, rhs, choice);
    slots[0].v.x = choice;
}
FUNCTION_PREAMBLE_LHS_RHS(min, float, f)
    slots[i_out] = fminf(lhs, rhs);
}
FUNCTION_PREAMBLE_LHS_RHS(min, Interval, i)
    uint8_t choice = 0;
    slots[i_out] = min(lhs, rhs, choice);
    slots[0].v.x = choice;
}

FUNCTION_PREAMBLE_LHS_IMM(max, float, f)
    slots[i_out] = fmaxf(lhs, rhs);
}
FUNCTION_PREAMBLE_LHS_IMM(max, Interval, i)
    uint8_t choice = 0;
    slots[i_out] = max(lhs, rhs, choice);
    slots[0].v.x = choice;
}
FUNCTION_PREAMBLE_LHS_RHS(max, float, f)
    slots[i_out] = fmaxf(lhs, rhs);
}
FUNCTION_PREAMBLE_LHS_RHS(max, Interval, i)
    uint8_t choice = 0;
    slots[i_out] = max(lhs, rhs, choice);
    slots[0].v.x = choice;
}

#define COMMUTATIVE_OP(name, form)                                  \
FUNCTION_PREAMBLE_LHS_IMM(name, Interval, i)                        \
    slots[i_out] = (form);                                          \
}                                                                   \
FUNCTION_PREAMBLE_LHS_RHS(name, Interval, i)                        \
    slots[i_out] = (form);                                          \
}                                                                   \
FUNCTION_PREAMBLE_LHS_IMM(name, float, f)                           \
    slots[i_out] = (form);                                          \
}                                                                   \
FUNCTION_PREAMBLE_LHS_RHS(name, float, f)                           \
    slots[i_out] = (form);                                          \
}

COMMUTATIVE_OP(add, lhs + rhs);
COMMUTATIVE_OP(mul, lhs * rhs);

#define NONCOMMUTATIVE_OP(name, form)                               \
FUNCTION_PREAMBLE_IMM_RHS(name, Interval, i)                        \
    slots[i_out] = (form);                                          \
}                                                                   \
FUNCTION_PREAMBLE_IMM_RHS(name, float, f)                           \
    slots[i_out] = (form);                                          \
}                                                                   \
COMMUTATIVE_OP(name, form)

NONCOMMUTATIVE_OP(sub, lhs - rhs);
NONCOMMUTATIVE_OP(div, lhs / rhs);

#define UNARY_OP(name, form_f, form_i)                              \
FUNCTION_PREAMBLE_LHS(name, Interval, i)                            \
    slots[i_out] = (form_i);                                        \
}                                                                   \
FUNCTION_PREAMBLE_LHS(name, float, f)                               \
    slots[i_out] = (form_f);                                        \
}
#define UNARY_OP_F(func) UNARY_OP(func, func##f(lhs), func(lhs))

// Completely different shapes
UNARY_OP(abs, fabsf(lhs), abs(lhs))
UNARY_OP(square, lhs * lhs, square(lhs))

// Same form for float and interval
UNARY_OP(neg, -lhs, -lhs)

// Standardized names based on function
UNARY_OP_F(sqrt)
UNARY_OP_F(asin)
UNARY_OP_F(acos)
UNARY_OP_F(atan)
UNARY_OP_F(exp)
UNARY_OP_F(sin)
UNARY_OP_F(cos)
UNARY_OP_F(log)

////////////////////////////////////////////////////////////////////////////////
static inline __device__
int4 unpack(int32_t pos, int32_t tiles_per_side)
{
    return make_int4(pos % tiles_per_side,
                    (pos / tiles_per_side) % tiles_per_side,
                    (pos / tiles_per_side) / tiles_per_side,
                     pos % (tiles_per_side * tiles_per_side));
}

////////////////////////////////////////////////////////////////////////////////
__global__
void v3_preload_tiles(v3_tile_node_t* const __restrict__ in_tiles,
                      const int32_t in_tile_count,
                      const int32_t offset)
{
    const int32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count) {
        return;
    }

    in_tiles[tile_index].position = tile_index + offset;
    in_tiles[tile_index].tape = 0;
    in_tiles[tile_index].next = -1;
}

__global__
void v3_calculate_intervals(const v3_tile_node_t* const __restrict__ in_tiles,
                            const uint32_t in_tile_count,
                            const uint32_t tiles_per_side,
                            const Eigen::Matrix4f mat,
                            Interval* const __restrict__ values)
{
    const uint32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count) {
        return;
    }

    const int4 pos = unpack(in_tiles[tile_index].position, tiles_per_side);
    const Interval ix = {(pos.x / (float)tiles_per_side - 0.5f) * 2.0f,
                   ((pos.x + 1) / (float)tiles_per_side - 0.5f) * 2.0f};
    const Interval iy = {(pos.y / (float)tiles_per_side - 0.5f) * 2.0f,
                   ((pos.y + 1) / (float)tiles_per_side - 0.5f) * 2.0f};
    const Interval iz = {(pos.z / (float)tiles_per_side - 0.5f) * 2.0f,
                   ((pos.z + 1) / (float)tiles_per_side - 0.5f) * 2.0f};

    Interval ix_, iy_, iz_, iw_;
    ix_ = mat(0, 0) * ix +
          mat(0, 1) * iy +
          mat(0, 2) * iz + mat(0, 3);
    iy_ = mat(1, 0) * ix +
          mat(1, 1) * iy +
          mat(1, 2) * iz + mat(1, 3);
    iz_ = mat(2, 0) * ix +
          mat(2, 1) * iy +
          mat(2, 2) * iz + mat(2, 3);
    iw_ = mat(3, 0) * ix +
          mat(3, 1) * iy +
          mat(3, 2) * iz + mat(3, 3);

    // Projection!
    ix_ = ix_ / iw_;
    iy_ = iy_ / iw_;
    iz_ = iz_ / iw_;

    values[tile_index * 3] = ix_;
    values[tile_index * 3 + 1] = iy_;
    values[tile_index * 3 + 2] = iz_;
}

__global__
void v3_eval_tiles_i(uint64_t* const __restrict__ tape_data,
                     int32_t* const __restrict__ tape_index,
                     int32_t* const __restrict__ image,
                     const uint32_t tiles_per_side,

                     v3_tile_node_t* const __restrict__ in_tiles,
                     const int32_t in_tile_count,

                     const Interval* __restrict__ values)
{
    const int32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count) {
        return;
    }

    // Check to see if we're masked
    if (in_tiles[tile_index].position == -1) {
        return;
    }

    Interval slots[128];
    slots[((const uint8_t*)tape_data)[1]] = values[tile_index * 3];
    slots[((const uint8_t*)tape_data)[2]] = values[tile_index * 3 + 1];
    slots[((const uint8_t*)tape_data)[3]] = values[tile_index * 3 + 2];

    // Pick out the tape based on the pointer stored in the tiles list
    const uint64_t* __restrict__ data = &tape_data[in_tiles[tile_index].tape];

    uint32_t choices[128] = {0};
    int choice_index = 0;
    bool has_any_choice = false;

    while (OP(++data)) {
        switch (OP(data)) {
            case GPU_OP_JUMP: data += JUMP_TARGET(data); continue;

            case GPU_OP_SQUARE_LHS: square_lhs_i(*data, slots); break;
            case GPU_OP_SQRT_LHS: sqrt_lhs_i(*data, slots); break;
            case GPU_OP_NEG_LHS: neg_lhs_i(*data, slots); break;
            case GPU_OP_SIN_LHS: sin_lhs_i(*data, slots); break;
            case GPU_OP_COS_LHS: cos_lhs_i(*data, slots); break;
            case GPU_OP_ASIN_LHS: asin_lhs_i(*data, slots); break;
            case GPU_OP_ACOS_LHS: acos_lhs_i(*data, slots); break;
            case GPU_OP_ATAN_LHS: atan_lhs_i(*data, slots); break;
            case GPU_OP_EXP_LHS: exp_lhs_i(*data, slots); break;
            case GPU_OP_ABS_LHS: abs_lhs_i(*data, slots); break;
            case GPU_OP_LOG_LHS: log_lhs_i(*data, slots); break;

            // Commutative opcodes
            case GPU_OP_ADD_LHS_IMM: add_lhs_imm_i(*data, slots); break;
            case GPU_OP_ADD_LHS_RHS: add_lhs_rhs_i(*data, slots); break;
            case GPU_OP_MUL_LHS_IMM: mul_lhs_imm_i(*data, slots); break;
            case GPU_OP_MUL_LHS_RHS: mul_lhs_rhs_i(*data, slots); break;
            case GPU_OP_MIN_LHS_IMM: min_lhs_imm_i(*data, slots); break;
            case GPU_OP_MIN_LHS_RHS: min_lhs_rhs_i(*data, slots); break;
            case GPU_OP_MAX_LHS_IMM: max_lhs_imm_i(*data, slots); break;
            case GPU_OP_MAX_LHS_RHS: max_lhs_rhs_i(*data, slots); break;

            // Non-commutative opcodes
            case GPU_OP_SUB_LHS_IMM: sub_lhs_imm_i(*data, slots); break;
            case GPU_OP_SUB_IMM_RHS: sub_imm_rhs_i(*data, slots); break;
            case GPU_OP_SUB_LHS_RHS: sub_lhs_rhs_i(*data, slots); break;
            case GPU_OP_DIV_LHS_IMM: div_lhs_imm_i(*data, slots); break;
            case GPU_OP_DIV_IMM_RHS: div_imm_rhs_i(*data, slots); break;
            case GPU_OP_DIV_LHS_RHS: div_lhs_rhs_i(*data, slots); break;

            case GPU_OP_COPY_IMM: copy_imm_i(*data, slots); break;
            case GPU_OP_COPY_LHS: copy_lhs_i(*data, slots); break;
            case GPU_OP_COPY_RHS: copy_rhs_i(*data, slots); break;

            default: assert(false);
        }
        // If this opcode makes a choice, then append that choice to the list
        if (OP(data) >= GPU_OP_MIN_LHS_IMM && OP(data) <= GPU_OP_MAX_LHS_RHS) {
            const uint8_t c = slots[0].v.x;
            choices[choice_index / 16] |= (c << ((choice_index % 16) * 2));
            choice_index++;
            has_any_choice |= (c != 0);
        }
    }

    // Check the result
    const uint8_t i_out = I_OUT(data);
#if 0
    printf("%u:%u: [%f %f] [%f %f] [%f %f] => [%f %f]\n",
            blockIdx.x, threadIdx.x,
            values[tile_index * 3].lower(),
            values[tile_index * 3].upper(),
            values[tile_index * 3 + 1].lower(),
            values[tile_index * 3 + 1].upper(),
            values[tile_index * 3 + 2].lower(),
            values[tile_index * 3 + 2].upper(),
            slots[i_out].lower(),
            slots[i_out].upper());
#endif

    if (slots[i_out].lower() > 0.0f) {
        in_tiles[tile_index].position = -1;
        return;
    }

    // Masked
    const int4 pos = unpack(in_tiles[tile_index].position, tiles_per_side);
    if (image[pos.w] > pos.z) {
        in_tiles[tile_index].position = -1;
        return;
    }

    // Filled
    if (slots[i_out].upper() < 0.0f) {
        const int4 pos = unpack(in_tiles[tile_index].position, tiles_per_side);
        in_tiles[tile_index].position = -1;
        atomicMax(&image[pos.w], pos.z);
        return;
    }

    if (!has_any_choice) {
        return;
    }

    ////////////////////////////////////////////////////////////////////////////
    // Tape pushing!
    // Use this array to track which slots are active
    int* const __restrict__ active = (int*)slots;
    for (unsigned i=0; i < 128; ++i) {
        active[i] = false;
    }
    active[i_out] = true;

    // Claim a chunk of tape
    int32_t out_index = atomicAdd(tape_index, SUBTAPE_CHUNK_SIZE);
    int32_t out_offset = SUBTAPE_CHUNK_SIZE;
    assert(out_index + out_offset < NUM_SUBTAPES *
                                    SUBTAPE_CHUNK_SIZE);

    // Write out the end of the tape, which is the same as the ending
    // of the previous tape (0 opcode, with i_out as the last slot)
    out_offset--;
    tape_data[out_index + out_offset] = *data;

    while (OP(--data)) {
        const uint8_t op = OP(data);
        if (op == GPU_OP_JUMP) {
            data += JUMP_TARGET(data);
            continue;
        }

        const bool has_choice = op >= GPU_OP_MIN_LHS_IMM &&
                                op <= GPU_OP_MAX_LHS_RHS;
        choice_index -= has_choice;

        const uint8_t i_out = I_OUT(data);
        if (!active[i_out]) {
            continue;
        }

        assert(!has_choice || choice_index >= 0);

        const uint8_t choice = has_choice
            ? ((choices[choice_index / 16] >>
              ((choice_index % 16) * 2)) & 3)
            : 0;

        // If we're about to write a new piece of data to the tape,
        // (and are done with the current chunk), then we need to
        // add another link to the linked list.
        --out_offset;
        if (out_offset == 0) {
            const int32_t prev_index = out_index;
            out_index = atomicAdd(tape_index, SUBTAPE_CHUNK_SIZE);
            out_offset = SUBTAPE_CHUNK_SIZE;
            assert(out_index + out_offset < NUM_SUBTAPES *
                                            SUBTAPE_CHUNK_SIZE);
            --out_offset;

            // Forward-pointing link
            OP(&tape_data[out_index + out_offset]) = GPU_OP_JUMP;
            const int32_t delta = (int32_t)prev_index -
                                  (int32_t)(out_index + out_offset);
            JUMP_TARGET(&tape_data[out_index + out_offset]) = delta;

            // Backward-pointing link
            OP(&tape_data[prev_index]) = GPU_OP_JUMP;
            JUMP_TARGET(&tape_data[prev_index]) = -delta;

            // We've written the jump, so adjust the offset again
            --out_offset;
        }

        active[i_out] = false;
        tape_data[out_index + out_offset] = *data;
        if (choice == 0) {
            const uint8_t i_lhs = I_LHS(data);
            active[i_lhs] = true;
            const uint8_t i_rhs = I_RHS(data);
            active[i_rhs] = true;
        } else if (choice == 1 /* LHS */) {
            // The non-immediate is always the LHS in commutative ops, and
            // min/max (the only clauses that produce a choice) are commutative
            const uint8_t i_lhs = I_LHS(data);
            if (i_lhs == i_out) {
                ++out_offset;
            } else {
                OP(&tape_data[out_index + out_offset]) = GPU_OP_COPY_LHS;
            }
            active[i_lhs] = true;
        } else if (choice == 2 /* RHS */) {
            const uint8_t i_rhs = I_RHS(data);
            if (i_rhs) {
                if (i_rhs == i_out) {
                    ++out_offset;
                } else {
                    OP(&tape_data[out_index + out_offset]) = GPU_OP_COPY_RHS;
                }
                active[i_rhs] = true;
            } else {
                OP(&tape_data[out_index + out_offset]) = GPU_OP_COPY_IMM;
            }
        }
    }

    // Write the beginning of the tape
    out_offset--;
    tape_data[out_index + out_offset] = *data;

    // Record the beginning of the tape in the output tile
    in_tiles[tile_index].tape = out_index + out_offset;
}

////////////////////////////////////////////////////////////////////////////////

__global__
void v3_mask_filled_tiles(int32_t* const __restrict__ image,
                          const uint32_t tiles_per_side,

                          v3_tile_node_t* const __restrict__ in_tiles,
                          const int32_t in_tile_count)
{
    const int32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count) {
        return;
    }

    const int32_t tile = in_tiles[tile_index].position;
    // Already marked as filled or empty
    if (tile == -1) {
        return;
    }

    const int4 pos = unpack(tile, tiles_per_side);

    // If this tile is completely masked by the image, then skip it
    if (image[pos.w] > pos.z) {
        in_tiles[tile_index].position = -1;
    }
}

////////////////////////////////////////////////////////////////////////////////

// Sets the tile.next to an index in the upcoming tile list, without
// actually doing any work (since that list may not be allocated yet)
__global__
void v3_assign_next_nodes(v3_tile_node_t* const __restrict__ in_tiles,
                          const int32_t in_tile_count,

                          int32_t* __restrict__ const num_active_tiles)
{
    const int32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count) {
        return;
    }

    const bool is_active = tile_index < in_tile_count &&
                           in_tiles[tile_index].position != -1;

    // Do two levels of accumulation, to reduce atomic pressure on a single
    // global variable.  Does this help?  Who knows!
    __shared__ int local_offset;
    if (threadIdx.x == 0) {
        local_offset = 0;
    }
    __syncthreads();

    int my_offset;
    if (is_active) {
        my_offset = atomicAdd(&local_offset, 1);
    }
    __syncthreads();

    // Only one thread gets to contribute to the global offset
    if (threadIdx.x == 0) {
        local_offset = atomicAdd(num_active_tiles, local_offset);
    }
    __syncthreads();

    if (is_active) {
        in_tiles[tile_index].next = local_offset + my_offset;
    } else {
        in_tiles[tile_index].next = -1;
    }
}

// Copies each active tile into 64 subtiles
__global__
void v3_subdivide_active_tiles(
        const v3_tile_node_t* const __restrict__ in_tiles,
        const int32_t in_tile_count,
        const int32_t tiles_per_side,
        v3_tile_node_t* const __restrict__ out_tiles)
{
    const int32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count || in_tiles[tile_index].next == -1) {
        return;
    }

    const int t = in_tiles[tile_index].next * 64;
    const int4 pos = unpack(in_tiles[tile_index].position, tiles_per_side);
    const int32_t subtiles_per_side = tiles_per_side * 4;

    for (int i=0; i < 64; ++i) {
        const int4 sub = unpack(i, 4);
        const int32_t sx = pos.x * 4 + sub.x;
        const int32_t sy = pos.y * 4 + sub.y;
        const int32_t sz = pos.z * 4 + sub.z;
        const int32_t next_tile =
            sx +
            sy * subtiles_per_side +
            sz * subtiles_per_side * subtiles_per_side;

        out_tiles[t + i].position = next_tile;
        out_tiles[t + i].tape = in_tiles[tile_index].tape;
        out_tiles[t + i].next = -1;
    }
}

// Copies each active tile into the out_tiles list, clearing its `next` value.
// This is used right before per-pixel evaluation, which wants a compact list
// of active tiles, but doesn't need to subdivide them by 64 itself.
__global__
void v3_copy_active_tiles(v3_tile_node_t* const __restrict__ in_tiles,
                          const int32_t in_tile_count,
                          const int32_t tiles_per_side,
                          v3_tile_node_t* const __restrict__ out_tiles)
{
    const int32_t tile_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (tile_index >= in_tile_count || in_tiles[tile_index].next == -1) {
        return;
    }
    const int t = in_tiles[tile_index].next;
    out_tiles[t].position = in_tiles[tile_index].position;
    out_tiles[t].tape = in_tiles[tile_index].tape;
    out_tiles[t].next = -1;
    in_tiles[tile_index].next = -1;
}

////////////////////////////////////////////////////////////////////////////////

__global__
void v3_copy_filled(const int32_t* __restrict__ prev,
                    int32_t* __restrict__ image,
                    const int32_t image_size_px)
{
    const int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    const int32_t y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < image_size_px && y < image_size_px) {
        int32_t t = prev[x / 4 + y / 4 * (image_size_px / 4)];
        if (t) {
            image[x + y * image_size_px] = t * 4 + 3;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////

__global__
void v3_eval_voxels_f(const uint64_t* const __restrict__ tape_data,
                      int32_t* const __restrict__ image,
                      const uint32_t tiles_per_side,

                      v3_tile_node_t* const __restrict__ in_tiles,
                      const int32_t in_tile_count,
                      Eigen::Matrix4f mat)
{
    // Each tile is executed by 64 threads (one for each voxel).
    //
    // This is different from the v3_eval_tiles_i function, which evaluates one
    // tile per thread, because the tiles are already expanded by 64x by the
    // time they're stored in the in_tiles list.
    const int32_t voxel_index = threadIdx.x + blockIdx.x * blockDim.x;
    const int32_t tile_index = voxel_index / 64;
    if (tile_index >= in_tile_count) {
        return;
    }

    float slots[128];

    {   // Load values into registers, subdividing by 4x on each axis
        const int4 pos = unpack(in_tiles[tile_index].position, tiles_per_side);
        const int4 sub = unpack(threadIdx.x % 64, 4);

        const int32_t px = pos.x * 4 + sub.x;
        const int32_t py = pos.y * 4 + sub.y;
        const int32_t pz = pos.z * 4 + sub.z;

        // Early return if this pixel won't ever be filled
        if (image[px + py * tiles_per_side * 4] >= pz) {
            return;
        }

        const float size_recip = 1.0f / (tiles_per_side * 4);

        const float fx = ((px + 0.5f) * size_recip - 0.5f) * 2.0f;
        const float fy = ((py + 0.5f) * size_recip - 0.5f) * 2.0f;
        const float fz = ((pz + 0.5f) * size_recip - 0.5f) * 2.0f;

        // Otherwise, calculate the X/Y/Z values
        const float fw_ = mat(3, 0) * fx +
                          mat(3, 1) * fy +
                          mat(3, 2) * fz + mat(3, 3);
        for (unsigned i=0; i < 3; ++i) {
            slots[((const uint8_t*)tape_data)[i + 1]] =
                (mat(i, 0) * fx +
                 mat(i, 1) * fy +
                 mat(i, 2) * fz + mat(0, 3)) / fw_;
        }
    }

    // Pick out the tape based on the pointer stored in the tiles list
    const uint64_t* __restrict__ data = &tape_data[in_tiles[tile_index].tape];

    while (OP(++data)) {
        switch (OP(data)) {
            case GPU_OP_JUMP: data += JUMP_TARGET(data); continue;

            case GPU_OP_SQUARE_LHS: square_lhs_f(*data, slots); break;
            case GPU_OP_SQRT_LHS: sqrt_lhs_f(*data, slots); break;
            case GPU_OP_NEG_LHS: neg_lhs_f(*data, slots); break;
            case GPU_OP_SIN_LHS: sin_lhs_f(*data, slots); break;
            case GPU_OP_COS_LHS: cos_lhs_f(*data, slots); break;
            case GPU_OP_ASIN_LHS: asin_lhs_f(*data, slots); break;
            case GPU_OP_ACOS_LHS: acos_lhs_f(*data, slots); break;
            case GPU_OP_ATAN_LHS: atan_lhs_f(*data, slots); break;
            case GPU_OP_EXP_LHS: exp_lhs_f(*data, slots); break;
            case GPU_OP_ABS_LHS: abs_lhs_f(*data, slots); break;
            case GPU_OP_LOG_LHS: log_lhs_f(*data, slots); break;

            // Commutative opcodes
            case GPU_OP_ADD_LHS_IMM: add_lhs_imm_f(*data, slots); break;
            case GPU_OP_ADD_LHS_RHS: add_lhs_rhs_f(*data, slots); break;
            case GPU_OP_MUL_LHS_IMM: mul_lhs_imm_f(*data, slots); break;
            case GPU_OP_MUL_LHS_RHS: mul_lhs_rhs_f(*data, slots); break;
            case GPU_OP_MIN_LHS_IMM: min_lhs_imm_f(*data, slots); break;
            case GPU_OP_MIN_LHS_RHS: min_lhs_rhs_f(*data, slots); break;
            case GPU_OP_MAX_LHS_IMM: max_lhs_imm_f(*data, slots); break;
            case GPU_OP_MAX_LHS_RHS: max_lhs_rhs_f(*data, slots); break;

            // Non-commutative opcodes
            case GPU_OP_SUB_LHS_IMM: sub_lhs_imm_f(*data, slots); break;
            case GPU_OP_SUB_IMM_RHS: sub_imm_rhs_f(*data, slots); break;
            case GPU_OP_SUB_LHS_RHS: sub_lhs_rhs_f(*data, slots); break;
            case GPU_OP_DIV_LHS_IMM: div_lhs_imm_f(*data, slots); break;
            case GPU_OP_DIV_IMM_RHS: div_imm_rhs_f(*data, slots); break;
            case GPU_OP_DIV_LHS_RHS: div_lhs_rhs_f(*data, slots); break;

            case GPU_OP_COPY_IMM: copy_imm_f(*data, slots); break;
            case GPU_OP_COPY_LHS: copy_lhs_f(*data, slots); break;
            case GPU_OP_COPY_RHS: copy_rhs_f(*data, slots); break;
        }
    }

    // Check the result
    const uint8_t i_out = I_OUT(data);
    if (slots[i_out] < 0.0f) {
        const int4 pos = unpack(in_tiles[tile_index].position, tiles_per_side);
        const int4 sub = unpack(threadIdx.x % 64, 4);
        const int32_t px = pos.x * 4 + sub.x;
        const int32_t py = pos.y * 4 + sub.y;
        const int32_t pz = pos.z * 4 + sub.z;

        atomicMax(&image[px + py * tiles_per_side * 4], pz);
    }
}


////////////////////////////////////////////////////////////////////////////////

v3_blob_t build_v3_blob(libfive::Tree tree, const int32_t image_size_px) {
    v3_blob_t out = {0};

    for (unsigned i=0; i < 4; ++i) {
        const unsigned tile_size_px = 64 / (1 << (i * 2));
        out.stages[i].filled = CUDA_MALLOC(
                int32_t,
                pow(image_size_px / tile_size_px, 2));
    }

    out.image_size_px = image_size_px;

    out.tape_data = CUDA_MALLOC(uint64_t, NUM_SUBTAPES * SUBTAPE_CHUNK_SIZE);
    out.tape_index = CUDA_MALLOC(int32_t, 1);
    *out.tape_index = 0;

    // Allocate an index to keep track of active tiles
    out.num_active_tiles = CUDA_MALLOC(int32_t, 1);

    // Allocate a bunch of scratch space for passing intervals around
    out.values = CUDA_MALLOC(Interval, NUM_THREADS * NUM_BLOCKS * 3);

    // The first array of tiles must have enough space to hold all of the
    // 64^3 tiles in the volume, which shouldn't be too much.
    out.stages[0].tiles = CUDA_MALLOC(
            v3_tile_node_t,
            pow(out.image_size_px / 64, 3));

    // We leave the other stage_t's input/output arrays unallocated for now,
    // since they're initialized to all zeros and will be resized to fit later.

    // TAPE PLANNING TIME!
    // Hold a single cache lock to avoid needing mutex locks everywhere
    auto lock = libfive::Cache::instance();

    auto ordered = tree.orderedDfs();

    std::map<libfive::Tree::Id, libfive::Tree::Id> last_used;
    bool axes_used[3] = {false, false, false};
    for (auto& c : ordered) {
        if (c->op != libfive::Opcode::CONSTANT) {
            // Very simple tracking of active spans, without clause reordering
            // or any other cleverness.
            last_used[c.lhs().id()] = c.id();
            last_used[c.rhs().id()] = c.id();
        }
        axes_used[0] |= c == libfive::Tree::X();
        axes_used[1] |= c == libfive::Tree::Y();
        axes_used[2] |= c == libfive::Tree::Z();
    }

    std::vector<uint8_t> free_slots;
    std::map<libfive::Tree::Id, uint8_t> bound_slots;
    uint8_t num_slots = 1;

    auto getSlot = [&](libfive::Tree::Id id) {
        // Pick a slot for the output of this opcode
        uint8_t out;
        if (free_slots.size()) {
            out = free_slots.back();
            free_slots.pop_back();
        } else {
            out = num_slots++;
            if (num_slots == UINT8_MAX) {
                fprintf(stderr, "Ran out of slots!\n");
            }
        }
        bound_slots[id] = out;
        return out;
    };

    // Bind the axes to known slots, so that we can store their values
    // before beginning an evaluation.
    const libfive::Tree axis_trees[3] = {
        libfive::Tree::X(),
        libfive::Tree::Y(),
        libfive::Tree::Z()};
    uint64_t start = 0;
    for (unsigned i=0; i < 3; ++i) {
        if (axes_used[i]) {
            ((uint8_t*)&start)[i + 1] = getSlot(axis_trees[i].id());
        }
    }
    std::vector<uint64_t> flat;
    flat.reserve(ordered.size());
    flat.push_back(start);

    auto get_reg = [&](const std::shared_ptr<libfive::Tree::Tree_>& tree) {
        auto itr = bound_slots.find(tree.get());
        if (itr != bound_slots.end()) {
            return itr->second;
        } else {
            fprintf(stderr, "Could not find bound slots");
            return static_cast<uint8_t>(0);
        }
    };

    for (auto& c : ordered) {
        uint64_t clause = 0;
        switch (c->op) {
            using namespace libfive::Opcode;

            case CONSTANT:
            case VAR_X:
            case VAR_Y:
            case VAR_Z:
                continue;

#define OP_UNARY(p) \
            case OP_##p: { \
                OP(&clause) = GPU_OP_##p##_LHS;      \
                I_LHS(&clause) = get_reg(c->lhs);    \
                break;                              \
            }
            OP_UNARY(SQUARE)
            OP_UNARY(SQRT);
            OP_UNARY(NEG);
            OP_UNARY(SIN);
            OP_UNARY(COS);
            OP_UNARY(ASIN);
            OP_UNARY(ACOS);
            OP_UNARY(ATAN);
            OP_UNARY(EXP);
            OP_UNARY(ABS);
            OP_UNARY(LOG);

#define OP_COMMUTATIVE(p) \
            case OP_##p: { \
                if (c->lhs->op == CONSTANT) {                   \
                    OP(&clause) = GPU_OP_##p##_LHS_IMM;         \
                    I_LHS(&clause) = get_reg(c->rhs);           \
                    IMM(&clause) = c->lhs->value;               \
                } else if (c->rhs->op == CONSTANT) {            \
                    OP(&clause) = GPU_OP_##p##_LHS_IMM;         \
                    I_LHS(&clause) = get_reg(c->lhs);           \
                    IMM(&clause) = c->rhs->value;               \
                } else {                                        \
                    OP(&clause) = GPU_OP_##p##_LHS_RHS;         \
                    I_LHS(&clause) = get_reg(c->lhs);           \
                    I_RHS(&clause) = get_reg(c->rhs);           \
                }                                               \
                break;                                          \
            }
            OP_COMMUTATIVE(ADD)
            OP_COMMUTATIVE(MUL)
            OP_COMMUTATIVE(MIN)
            OP_COMMUTATIVE(MAX)

#define OP_NONCOMMUTATIVE(p) \
            case OP_##p: { \
                if (c->lhs->op == CONSTANT) {                   \
                    OP(&clause) = GPU_OP_##p##_IMM_RHS;         \
                    I_RHS(&clause) = get_reg(c->rhs);           \
                    IMM(&clause) = c->lhs->value;               \
                } else if (c->rhs->op == CONSTANT) {            \
                    OP(&clause) = GPU_OP_##p##_LHS_IMM;         \
                    I_LHS(&clause) = get_reg(c->lhs);           \
                    IMM(&clause) = c->rhs->value;               \
                } else {                                        \
                    OP(&clause) = GPU_OP_##p##_LHS_RHS;         \
                    I_LHS(&clause) = get_reg(c->lhs);           \
                    I_RHS(&clause) = get_reg(c->rhs);           \
                }                                               \
                break;                                          \
            }
            OP_NONCOMMUTATIVE(SUB)
            OP_NONCOMMUTATIVE(DIV)

            case INVALID:
            case OP_TAN:
            case OP_RECIP:
            case OP_ATAN2:
            case OP_POW:
            case OP_NTH_ROOT:
            case OP_MOD:
            case OP_NANFILL:
            case OP_COMPARE:
            case VAR_FREE:
            case CONST_VAR:
            case ORACLE:
            case LAST_OP:
                fprintf(stderr, "Unimplemented opcode");
                break;
        }

        // Release slots if this was their last use.  We do this now so
        // that one of them can be reused for the output slots below.
        for (auto& h : {c.lhs().id(), c.rhs().id()}) {
            if (h != nullptr &&
                h->op != libfive::Opcode::CONSTANT &&
                last_used[h] == c.id())
            {
                auto itr = bound_slots.find(h);
                free_slots.push_back(itr->second);
                bound_slots.erase(itr);
            }
        }

        I_OUT(&clause) = getSlot(c.id());
        flat.push_back(clause);
    }
    {   // Push the end of the tape, which points to the final clauses's
        // output slot so that we know where to read the result.
        uint64_t end = 0;
        I_OUT(&end) = get_reg(ordered.back().operator->());
        flat.push_back(end);
    }

    CUDA_CHECK(hipMemcpy(out.tape_data, flat.data(),
                          sizeof(uint64_t) * flat.size(),
                          hipMemcpyHostToDevice));
    out.tape_length = flat.size();

    return out;
}

void free_v3_blob(v3_blob_t& blob) {
    for (unsigned i=0; i < 4; ++i) {
        CUDA_FREE(blob.stages[i].filled);
        CUDA_FREE(blob.stages[i].tiles);
    }

    CUDA_FREE(blob.tape_data);
    CUDA_FREE(blob.tape_index);

    CUDA_FREE(blob.num_active_tiles);

    CUDA_FREE(blob.values);
}

////////////////////////////////////////////////////////////////////////////////

void render_v3_blob(v3_blob_t& blob, Eigen::Matrix4f mat) {
    // Reset the tape index
    *blob.tape_index = blob.tape_length;

    ////////////////////////////////////////////////////////////////////////////
    // Evaluation of 64x64x64 tiles
    ////////////////////////////////////////////////////////////////////////////

    // Reset all of the data arrays
    for (unsigned i=0; i < 4; ++i) {
        const unsigned tile_size_px = 64 / (1 << (i * 2));
        CUDA_CHECK(hipMemset(blob.stages[i].filled, 0, sizeof(int32_t) *
                              pow(blob.image_size_px / tile_size_px, 2)));
    }

    // Go the whole list of first-stage tiles, assigning each to
    // be [position, tape = 0, next = -1]
    unsigned stride = NUM_BLOCKS * NUM_THREADS;
    unsigned count = pow(blob.image_size_px / 64, 3);
    for (unsigned offset=0; offset < count; offset += stride) {
        v3_preload_tiles<<<NUM_BLOCKS, NUM_THREADS>>>(
            blob.stages[0].tiles + offset,
            std::min(stride, count - offset),
            offset);
    }

    // Iterate over 64^3, 16^3, 4^3 tiles
    for (unsigned i=0; i < 3; ++i) {
        //printf("BEGINNING STAGE %u\n", i);
        const unsigned tile_size_px = 64 / (1 << (i * 2));

        // Now loop through doing evaluation, one batch at a time
        for (unsigned offset=0; offset < count; offset += stride) {
            /*
            printf("    looping with offset = %u, stride = %u, count = %u\n",
                    offset, stride, count);
            */
            const int active_threads = std::min(stride, count - offset);
            const int active_blocks = (active_threads + NUM_THREADS - 1) / NUM_THREADS;

            // Unpack position values into interval X/Y/Z in the values array
            // This is done in a separate kernel to avoid bloating the
            // eval_tiles_i kernel with more registers, which is detrimental
            // to occupancy.
            v3_calculate_intervals<<<active_blocks, NUM_THREADS>>>(
                blob.stages[i].tiles + offset,
                active_threads,
                blob.image_size_px / tile_size_px,
                mat,
                (Interval*)blob.values);

            // Mark every tile which is covered in the image as masked,
            // which means it will be skipped later on.  We do this again below,
            // but it's basically free, so we should do it here and simplify
            // the logic in eval_tiles_i.
            v3_mask_filled_tiles<<<active_blocks, NUM_THREADS>>>(
                blob.stages[i].filled,
                blob.image_size_px / tile_size_px,
                blob.stages[i].tiles + offset,
                active_threads);

            // Do the actual tape evaluation, which is the expensive step
            v3_eval_tiles_i<<<active_blocks, NUM_THREADS>>>(
                blob.tape_data,
                blob.tape_index,
                blob.stages[i].filled,
                blob.image_size_px / tile_size_px,

                blob.stages[i].tiles + offset,
                active_threads,

                (Interval*)blob.values);
        }

        // Mark the total number of active tiles (from this stage) to 0
        hipMemsetAsync(blob.num_active_tiles, 0, sizeof(int32_t));

        // Now that we have evaluated every tile at this level, we do one more
        // round of occlusion culling before accumulating tiles to render at
        // the next phase.
        for (unsigned offset=0; offset < count; offset += stride) {
            const int active_threads = std::min(stride, count - offset);
            const int active_blocks = (active_threads + NUM_THREADS - 1) / NUM_THREADS;

            v3_mask_filled_tiles<<<active_blocks, NUM_THREADS>>>(
                blob.stages[i].filled,
                blob.image_size_px / tile_size_px,
                blob.stages[i].tiles + offset,
                active_threads);

            // Count up active tiles, to figure out how much memory needs to be
            // allocated in the next stage.
            v3_assign_next_nodes<<<active_blocks, NUM_THREADS>>>(
                blob.stages[i].tiles + offset,
                active_threads,
                blob.num_active_tiles);
        }

        // Count the number of active tiles, which have been accumulated
        // through repeated calls to v3_assign_next_nodes
        int32_t num_active_tiles;
        hipMemcpy(&num_active_tiles, blob.num_active_tiles, sizeof(int32_t),
                   hipMemcpyDeviceToHost);
        if (i < 2) {
            num_active_tiles *= 64;
        }

        // Make sure that the subtiles buffer has enough room
        if (num_active_tiles > blob.stages[i + 1].tile_array_size) {
            blob.stages[i + 1].tile_array_size = num_active_tiles;
            CUDA_FREE(blob.stages[i + 1].tiles);
            blob.stages[i + 1].tiles = CUDA_MALLOC(
                    v3_tile_node_t, num_active_tiles);
        }

        if (i < 2) {
            // Build the new tile list from active tiles in the previous list
            for (unsigned offset=0; offset < count; offset += stride) {
                const int active_threads = std::min(stride, count - offset);
                const int active_blocks = (active_threads + NUM_THREADS - 1) / NUM_THREADS;

                v3_subdivide_active_tiles<<<active_blocks, NUM_THREADS>>>(
                    blob.stages[i].tiles + offset,
                    active_threads,
                    blob.image_size_px / tile_size_px,
                    blob.stages[i + 1].tiles);
            }
        } else {
            // Special case for per-pixel evaluation, which
            // doesn't unpack every single pixel (since that would take up
            // 64x extra space).
            for (unsigned offset=0; offset < count; offset += stride) {
                const int active_threads = std::min(stride, count - offset);
                const int active_blocks = (active_threads + NUM_THREADS - 1) / NUM_THREADS;

                v3_copy_active_tiles<<<active_blocks, NUM_THREADS>>>(
                    blob.stages[i].tiles + offset,
                    active_threads,
                    blob.image_size_px / tile_size_px,
                    blob.stages[i + 1].tiles);
            }
        }

        {   // Copy filled tiles into the next level's image (expanding them
            // by 64x).  This is cleaner that accumulating all of the levels
            // in a single pass, and could (possibly?) help with skipping
            // fully occluded tiles.
            const unsigned next_tile_size = tile_size_px / 4;
            const uint32_t u = ((blob.image_size_px / next_tile_size) / 32);
            v3_copy_filled<<<dim3(u + 1, u + 1), dim3(32, 32)>>>(
                    blob.stages[i].filled,
                    blob.stages[i + 1].filled,
                    blob.image_size_px / next_tile_size);
        }

        // Assign the next number of tiles to evaluate
        count = num_active_tiles;

        /*
        printf("------------------------------------------------------------\n");
        printf("Done with stage %u with %u tiles to do\n",
                i, count);
                */
    }

    // Time to render individual pixels!
    stride = NUM_BLOCKS * NUM_TILES;
    for (unsigned offset=0; offset < count; offset += stride) {
        //printf("Rendering pixels with offset %u, count %u\n", offset, count);
        const int active_tiles = std::min(stride, count - offset);
        const int active_blocks = (active_tiles*64 + NUM_THREADS - 1) / NUM_THREADS;

        v3_eval_voxels_f<<<active_blocks, NUM_THREADS>>>(
            blob.tape_data,
            blob.stages[3].filled,
            blob.image_size_px / 4,

            blob.stages[3].tiles + offset,
            active_tiles,

            mat);
    }
    CUDA_CHECK(hipDeviceSynchronize());
}

// END OF EXPERIMENTAL ZONE
////////////////////////////////////////////////////////////////////////////////
