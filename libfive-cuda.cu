#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// libfive
#include <libfive/tree/opcode.hpp>
#include <libfive/tree/tree.hpp>

// Our Interval arithmetic class
#include "gpu_interval.hpp"

struct Clause {
    const uint8_t opcode;
    const uint8_t banks;
    const uint16_t out;
    const uint16_t lhs;
    const uint16_t rhs;
};

// The Tape is an on-device representation, so the pointers
// are returned from hipMalloc.
struct Tape {
    const Clause* const __restrict__ tape;
    const uint32_t tape_length;

    const uint16_t num_regs;
    const uint16_t num_csg_choices;

    const float* const __restrict__ constants;
};

__device__ void walk(const Tape tape,
                     const Interval X, const Interval Y,
                     Interval* const __restrict__ regs,
                     uint8_t* const __restrict__ choices)
{
    uint32_t choice_index = 0;
    for (uint32_t i=0; i < tape.tape_length; ++i) {
        const Clause c = tape.tape[i];
#define LHS ((!(c.banks & 1) ? regs[c.lhs] : Interval{tape.constants[c.lhs], \
                                                     tape.constants[c.lhs]}))
#define RHS ((!(c.banks & 2) ? regs[c.rhs] : Interval{tape.constants[c.rhs], \
                                                     tape.constants[c.rhs]}))
        using namespace libfive::Opcode;
        switch (c.opcode) {
            case VAR_X: regs[c.out] = X; break;
            case VAR_Y: regs[c.out] = Y; break;

            case OP_SQUARE: regs[c.out] = LHS.square(); break;
            case OP_SQRT: regs[c.out] = LHS.sqrt(); break;
            case OP_NEG: regs[c.out] = -LHS; break;
            // Skipping transcendental functions for now

            case OP_ADD: regs[c.out] = LHS + RHS; break;
            case OP_MUL: regs[c.out] = LHS * RHS; break;
            case OP_MIN: if (LHS.upper < RHS.lower) {
                             choices[choice_index] = 1;
                             regs[c.out] = LHS;
                         } else if (RHS.upper < LHS.lower) {
                             choices[choice_index] = 2;
                             regs[c.out] = RHS;
                         } else {
                             choices[choice_index] = 0;
                             regs[c.out] = LHS.min(RHS);
                         }
                         choice_index++;
                         break;
            case OP_MAX: if (LHS.lower > RHS.upper) {
                             choices[choice_index] = 1;
                             regs[c.out] = LHS;
                         } else if (RHS.lower > LHS.upper) {
                             choices[choice_index] = 2;
                             regs[c.out] = RHS;
                         } else {
                             choices[choice_index] = 0;
                             regs[c.out] = LHS.max(RHS);
                         }
                         choice_index++;
                         break;
            case OP_SUB: regs[c.out] = LHS - RHS; break;

            // Skipping various hard functions here
            default: break;
        }
    }
#undef LHS
#undef RHS
}

struct Output {
    uint32_t* const __restrict__ tiles;
    const uint32_t tiles_length;

    uint32_t num_active;
    uint32_t num_filled;
};

const static uint32_t IMAGE_SIZE_PX = 65535;
const static uint32_t TILE_SIZE_PX = 16;
const static uint32_t TILE_COUNT = IMAGE_SIZE_PX / TILE_SIZE_PX;
const static uint32_t TOTAL_TILES = TILE_COUNT * TILE_COUNT;

const static uint32_t NUM_BLOCKS = 128;
const static uint32_t THREADS_PER_BLOCK = TILE_COUNT / NUM_BLOCKS;

__global__ void processTiles(const Tape tape,
        // Flat array for all pseudoregisters
        Interval* const __restrict__ regs_,

        // Flat array for all CSG choices
        uint8_t* const __restrict__ csg_choices_,

        // Output data
        Output* const __restrict__ out)
{
    const float x = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    const float y = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;

    const Interval X = {x / TILE_COUNT, (x + 1) / TILE_COUNT};
    const Interval Y = {y / TILE_COUNT, (y + 1) / TILE_COUNT};

    // Unpack a 1D offset into the data arrays
    const uint32_t index = x * TILE_COUNT + y;
    auto regs = regs_ + index * tape.num_regs;
    auto csg_choices = csg_choices_ + index * tape.num_csg_choices;
    walk(tape, X, Y, regs, csg_choices);

    const Interval result = regs[tape.tape[tape.tape_length - 1].out];
    if (result.lower <= 0.0f && result.upper >= 0.0f) {
        printf("[%f %f][%f %f]: [%f %f]\n",
                X.lower, X.upper,
                Y.lower, Y.upper,
                result.lower, result.upper);
    }
}

Tape prepareTape(libfive::Tree tree) {
    auto ordered = tree.ordered();

    std::map<libfive::Tree::Id, libfive::Tree::Id> last_used;
    std::vector<float> constant_data;
    std::map<libfive::Tree::Id, uint16_t> constants;
    uint16_t num_csg_choices = 0;
    for (auto& c : ordered) {
        if (c->op == libfive::Opcode::CONSTANT) {
            // Store constants in a separate list
            constants.insert({c.id(), constant_data.size()});
            constant_data.push_back(c->value);
        } else {
            // Very simple tracking of active spans, without clause reordering
            // or any other cleverness.
            last_used.insert({c.lhs().id(), c.id()});
            last_used.insert({c.rhs().id(), c.id()});

            num_csg_choices += (c->op == libfive::Opcode::OP_MIN ||
                                c->op == libfive::Opcode::OP_MAX);
        }
    }

    std::list<uint16_t> free_registers;
    std::map<libfive::Tree::Id, uint16_t> bound_registers;
    uint16_t num_registers = 0;
    std::vector<Clause> flat;
    for (auto& c : ordered) {
        // Constants are not inserted into the tape, because they
        // live in a separate data array addressed with flags in
        // the 'banks' argument of a Clause.
        if (constants.find(c.id()) != constants.end()) {
            continue;
        }

        // Pick a registers for the output of this opcode
        uint16_t out;
        if (free_registers.size()) {
            out = free_registers.back();
            free_registers.pop_back();
        } else {
            out = num_registers++;
            if (num_registers == UINT16_MAX) {
                fprintf(stderr, "Ran out of registers!\n");
            }
        }
        bound_registers.insert({c.id(), out});

        uint8_t banks = 0;
        auto f = [&](libfive::Tree::Id id, uint8_t mask) {
            if (id == nullptr) {
                return static_cast<uint16_t>(0);
            }
            {   // Check whether this is a constant
                auto itr = constants.find(id);
                if (itr != constants.end()) {
                    banks |= mask;
                    return itr->second;
                }
            }
            {   // Otherwise, it must be a bound register
                auto itr = bound_registers.find(id);
                if (itr != bound_registers.end()) {
                    return itr->second;
                } else {
                    fprintf(stderr, "Could not LHS?");
                    return static_cast<uint16_t>(0);
                }
            }
        };

        const uint16_t lhs = f(c.lhs().id(), 1);
        const uint16_t rhs = f(c.rhs().id(), 2);

        flat.push_back({static_cast<uint8_t>(c->op), banks, out, lhs, rhs});

        std::cout << libfive::Opcode::toString(c->op) << " "
                  << ((banks & 1) ? constant_data[lhs] : lhs) << " "
                  << ((banks & 2) ? constant_data[rhs] : rhs) << " -> " << out << "\n";

        // Release registers if this was their last use
        for (auto& h : {c.lhs().id(), c.rhs().id()}) {
            if (h != nullptr && h->op != libfive::Opcode::CONSTANT &&
                last_used[h] == c.id())
            {
                auto itr = bound_registers.find(h);
                free_registers.push_back(itr->second);
                bound_registers.erase(itr);
            }
        }
    }

    Clause* d_flat_tape;
    checkCudaErrors(hipMalloc(
                reinterpret_cast<void **>(&d_flat_tape),
                sizeof(Clause) * flat.size()));
    checkCudaErrors(hipMemcpy(d_flat_tape, flat.data(),
                sizeof(Clause) * flat.size(),
                hipMemcpyHostToDevice));

    float* d_flat_constants;
    checkCudaErrors(hipMalloc(
                reinterpret_cast<void **>(&d_flat_constants),
                sizeof(float) * constant_data.size()));
    checkCudaErrors(hipMemcpy(
                d_flat_constants, constant_data.data(),
                sizeof(float) * constant_data.size(),
                hipMemcpyHostToDevice));

    return Tape {
        d_flat_tape,
        static_cast<uint32_t>(flat.size()),
        num_registers,
        num_csg_choices,
        d_flat_constants
    };
}

void callProcessTiles(Tape tape) {
    Interval* d_regs;
    checkCudaErrors(hipMalloc(
                reinterpret_cast<void **>(&d_regs),
                sizeof(Interval) * tape.num_regs * TOTAL_TILES));

    uint8_t* d_csg_choices;
    checkCudaErrors(hipMalloc(
                reinterpret_cast<void **>(&d_csg_choices),
                sizeof(uint8_t) * tape.num_csg_choices * TOTAL_TILES));

    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 grid(NUM_BLOCKS, NUM_BLOCKS);
    processTiles <<< grid, threads >>>(tape,
        d_regs, d_csg_choices,
        nullptr  /* out */);
    const auto code = hipGetLastError();
    if (code != hipSuccess) {
        fprintf(stderr, "Failed to launch: %s\n",
                hipGetErrorString(code));
    }
    hipDeviceSynchronize();
}

/**
 * Program main
 */
int main(int argc, char **argv)
{
    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    {   // CUDA, help me pick magic numbers:
        int min_grid_size;
        int block_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
            processTiles);
        printf("Min grid size: %i\tBlock size: %i\n", min_grid_size, block_size);
    }

    {
        auto X = libfive::Tree::X();
        auto Y = libfive::Tree::Y();
        auto circle = sqrt(X*X + Y*Y) - 1.0;
        auto tape = prepareTape(circle);
        callProcessTiles(tape);
    }
    return 0;
}
