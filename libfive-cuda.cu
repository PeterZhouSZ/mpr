#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// libfive
#include <libfive/tree/opcode.hpp>
#include <libfive/tree/tree.hpp>

// Our Interval arithmetic class
#include "gpu_interval.hpp"

struct Clause {
    const uint8_t opcode;
    const uint8_t banks;
    const uint16_t out;
    const uint16_t lhs;
    const uint16_t rhs;
};

// The Tape is an on-device representation, so the pointers
// are returned from hipMalloc.
struct Tape {
    const Clause* const __restrict__ tape;
    const uint32_t tape_length;

    const uint16_t num_regs;
    const uint16_t num_csg_choices;

    const float* const __restrict__ constants;
};

__device__ void walk(const Tape tape,
                     const Interval X, const Interval Y,
                     Interval* const __restrict__ regs,
                     uint8_t* const __restrict__ choices)
{
    uint32_t choice_index = 0;
    for (uint32_t i=0; i < tape.tape_length; ++i) {
        const Clause c = tape.tape[i];
#define LHS ((!(c.banks & 1) ? regs[c.lhs] : Interval{tape.constants[c.lhs], \
                                                     tape.constants[c.lhs]}))
#define RHS ((!(c.banks & 2) ? regs[c.rhs] : Interval{tape.constants[c.rhs], \
                                                     tape.constants[c.rhs]}))
        using namespace libfive::Opcode;
        switch (c.opcode) {
            case VAR_X: regs[c.out] = X; break;
            case VAR_Y: regs[c.out] = Y; break;

            case OP_SQUARE: regs[c.out] = LHS.square(); break;
            case OP_SQRT: regs[c.out] = LHS.sqrt(); break;
            case OP_NEG: regs[c.out] = -LHS; break;
            // Skipping transcendental functions for now

            case OP_ADD: regs[c.out] = LHS + RHS; break;
            case OP_MUL: regs[c.out] = LHS * RHS; break;
            case OP_MIN: if (LHS.upper < RHS.lower) {
                             choices[choice_index] = 1;
                             regs[c.out] = LHS;
                         } else if (RHS.upper < LHS.lower) {
                             choices[choice_index] = 2;
                             regs[c.out] = RHS;
                         } else {
                             choices[choice_index] = 0;
                             regs[c.out] = LHS.min(RHS);
                         }
                         choice_index++;
                         break;
            case OP_MAX: if (LHS.lower > RHS.upper) {
                             choices[choice_index] = 1;
                             regs[c.out] = LHS;
                         } else if (RHS.lower > LHS.upper) {
                             choices[choice_index] = 2;
                             regs[c.out] = RHS;
                         } else {
                             choices[choice_index] = 0;
                             regs[c.out] = LHS.max(RHS);
                         }
                         choice_index++;
                         break;
            case OP_SUB: regs[c.out] = LHS - RHS; break;

            // Skipping various hard functions here
            default: break;
        }
    }
#undef LHS
#undef RHS
}

struct Output {
    uint32_t* const __restrict__ tiles;
    const uint32_t tiles_length;

    uint32_t num_active;
    uint32_t num_filled;
};

__global__ void processTiles(const Tape tape,
        // Flat array for all pseudoregisters
        Interval* const __restrict__ regs_,

        // Flat array for all CSG choices
        uint8_t* const __restrict__ csg_choices_,

        // Output data
        Output* const __restrict__ out)
{
    assert(blockDim.x == blockDim.y);
    assert(gridDim.x == gridDim.y);

    const float x = blockIdx.x * blockDim.x + threadIdx.x;
    const float y = blockIdx.y * blockDim.y + threadIdx.y;

    const uint32_t TILE_COUNT = gridDim.x * blockDim.x;

    const Interval X = {x / TILE_COUNT, (x + 1) / TILE_COUNT};
    const Interval Y = {y / TILE_COUNT, (y + 1) / TILE_COUNT};

    // Unpack a 1D offset into the data arrays
    const uint32_t index = x * TILE_COUNT + y;
    auto regs = regs_ + index * tape.num_regs;
    auto csg_choices = csg_choices_ + index * tape.num_csg_choices;
    walk(tape, X, Y, regs, csg_choices);

    const Interval result = regs[tape.tape[tape.tape_length - 1].out];
    // If this tile is unambiguously filled, then mark it at the end
    // of the tiles list
    if (result.upper < 0.0f) {
        uint32_t i = atomicAdd(&out->num_filled, 1);
        out->tiles[out->tiles_length - 1 - i] = index;
    }
    // If the tile is ambiguous, then record it as needing further refinement
    else if (result.lower <= 0.0f && result.upper >= 0.0f) {
        uint32_t i = atomicAdd(&out->num_active, 2);
        out->tiles[i] = index;
        out->tiles[i + 1] = 0; // This will eventually be a subtape pointer
        printf("[%f %f][%f %f]: [%f %f]\n",
                X.lower, X.upper,
                Y.lower, Y.upper,
                result.lower, result.upper);
    }
}

template <unsigned TILE_COUNT>
__global__ void fillTiles(Output* const __restrict__ out,
                          uint8_t* __restrict__ image,
                          uint32_t* __restrict__ index)
{
    // We assume one thread per pixel in a tile
    const uint32_t TILE_SIZE_PX = blockDim.x;
    assert(blockDim.x == blockDim.y);

    const uint32_t dx = threadIdx.x;
    const uint32_t dy = threadIdx.y;

    const uint32_t num_active = out->num_active;
    while (1) {
        // The 0th thread in the block gets to pick out the index of
        // the target tile from the master list.
        __shared__ int i;
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            i = atomicAdd(index, 1);
        }
        __syncthreads();
        if (i >= num_active) {
            break;
        }

        // Pick a tile from the list
        const uint32_t tile = out->tiles[out->tiles_length - i - 1];

        // Convert from tile position to pixels
        const uint32_t px = (tile / TILE_COUNT) * TILE_SIZE_PX + dx;
        const uint32_t py = (tile % TILE_COUNT) * TILE_SIZE_PX + dy;

        image[px + py * TILE_SIZE_PX * TILE_COUNT] = 1;
    }
}

Tape prepareTape(libfive::Tree tree) {
    auto ordered = tree.ordered();

    std::map<libfive::Tree::Id, libfive::Tree::Id> last_used;
    std::vector<float> constant_data;
    std::map<libfive::Tree::Id, uint16_t> constants;
    uint16_t num_csg_choices = 0;
    for (auto& c : ordered) {
        if (c->op == libfive::Opcode::CONSTANT) {
            // Store constants in a separate list
            if (constant_data.size() == UINT16_MAX) {
                fprintf(stderr, "Ran out of constants!\n");
            }
            constants.insert({c.id(), constant_data.size()});
            constant_data.push_back(c->value);
        } else {
            // Very simple tracking of active spans, without clause reordering
            // or any other cleverness.
            last_used.insert({c.lhs().id(), c.id()});
            last_used.insert({c.rhs().id(), c.id()});

            num_csg_choices += (c->op == libfive::Opcode::OP_MIN ||
                                c->op == libfive::Opcode::OP_MAX);
        }
    }

    std::list<uint16_t> free_registers;
    std::map<libfive::Tree::Id, uint16_t> bound_registers;
    uint16_t num_registers = 0;
    std::vector<Clause> flat;
    for (auto& c : ordered) {
        // Constants are not inserted into the tape, because they
        // live in a separate data array addressed with flags in
        // the 'banks' argument of a Clause.
        if (constants.find(c.id()) != constants.end()) {
            continue;
        }

        // Pick a registers for the output of this opcode
        uint16_t out;
        if (free_registers.size()) {
            out = free_registers.back();
            free_registers.pop_back();
        } else {
            out = num_registers++;
            if (num_registers == UINT16_MAX) {
                fprintf(stderr, "Ran out of registers!\n");
            }
        }
        bound_registers.insert({c.id(), out});

        uint8_t banks = 0;
        auto f = [&](libfive::Tree::Id id, uint8_t mask) {
            if (id == nullptr) {
                return static_cast<uint16_t>(0);
            }
            {   // Check whether this is a constant
                auto itr = constants.find(id);
                if (itr != constants.end()) {
                    banks |= mask;
                    return itr->second;
                }
            }
            {   // Otherwise, it must be a bound register
                auto itr = bound_registers.find(id);
                if (itr != bound_registers.end()) {
                    return itr->second;
                } else {
                    fprintf(stderr, "Could not find bound register");
                    return static_cast<uint16_t>(0);
                }
            }
        };

        const uint16_t lhs = f(c.lhs().id(), 1);
        const uint16_t rhs = f(c.rhs().id(), 2);

        flat.push_back({static_cast<uint8_t>(c->op), banks, out, lhs, rhs});

        std::cout << libfive::Opcode::toString(c->op) << " "
                  << ((banks & 1) ? constant_data[lhs] : lhs) << " "
                  << ((banks & 2) ? constant_data[rhs] : rhs) << " -> "
                  << out << "\n";

        // Release registers if this was their last use
        for (auto& h : {c.lhs().id(), c.rhs().id()}) {
            if (h != nullptr && h->op != libfive::Opcode::CONSTANT &&
                last_used[h] == c.id())
            {
                auto itr = bound_registers.find(h);
                free_registers.push_back(itr->second);
                bound_registers.erase(itr);
            }
        }
    }

    Clause* d_tape;
    checkCudaErrors(hipMallocManaged(
                reinterpret_cast<void **>(&d_tape),
                sizeof(Clause) * flat.size()));

    float* d_flat_constants;
    checkCudaErrors(hipMallocManaged(
                reinterpret_cast<void **>(&d_flat_constants),
                sizeof(float) * constant_data.size()));

    checkCudaErrors(hipDeviceSynchronize());
    memcpy(d_tape, flat.data(), sizeof(Clause) * flat.size());
    memcpy(d_flat_constants, constant_data.data(),
           sizeof(float) * constant_data.size());

    return Tape {
        d_tape,
        static_cast<uint32_t>(flat.size()),
        num_registers,
        num_csg_choices,
        d_flat_constants
    };
}

template <unsigned IMAGE_SIZE_PX=256, unsigned TILE_SIZE_PX=16>
Output* callProcessTiles(Tape tape) {
    constexpr unsigned TILE_COUNT = IMAGE_SIZE_PX / TILE_SIZE_PX;
    constexpr unsigned TOTAL_TILES = TILE_COUNT * TILE_COUNT;

    constexpr unsigned NUM_BLOCKS = 8;
    constexpr unsigned THREADS_PER_BLOCK = TILE_COUNT / NUM_BLOCKS;
    printf("threads per block: %u\n", THREADS_PER_BLOCK);

    Interval* d_regs;
    checkCudaErrors(hipMallocManaged(
                reinterpret_cast<void **>(&d_regs),
                sizeof(Interval) * tape.num_regs * TOTAL_TILES));

    uint8_t* d_csg_choices;
    checkCudaErrors(hipMallocManaged(
                reinterpret_cast<void **>(&d_csg_choices),
                max(1, tape.num_csg_choices) * TOTAL_TILES));

    uint32_t* d_tiles;
    checkCudaErrors(hipMallocManaged(
                reinterpret_cast<void **>(&d_tiles),
                sizeof(uint32_t) * 2 * TOTAL_TILES));

    Output* d_out;
    checkCudaErrors(hipMallocManaged(
                reinterpret_cast<void **>(&d_out),
                sizeof(Output)));

    checkCudaErrors(hipDeviceSynchronize());
    new (d_out) Output { d_tiles, TOTAL_TILES * 2, 0, 0 };

    {
        dim3 grid(NUM_BLOCKS, NUM_BLOCKS);
        dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
        printf("threads per block: %u\tnumber of blocks: %u\n",
                THREADS_PER_BLOCK, NUM_BLOCKS);

        processTiles <<< grid, threads >>>(tape, d_regs, d_csg_choices, d_out);
        const auto code = hipGetLastError();
        if (code != hipSuccess) {
            fprintf(stderr, "Failed to launch: %s\n",
                    hipGetErrorString(code));
        }
    }

    {
        dim3 threads(TILE_SIZE_PX, TILE_SIZE_PX);
        dim3 grid(16, 16);

        uint32_t* d_index;
        checkCudaErrors(hipMallocManaged(
                    (void**)&d_index, sizeof(uint32_t)));

        uint8_t* d_image;
        checkCudaErrors(hipMallocManaged(
                    (void**)&d_image, IMAGE_SIZE_PX * IMAGE_SIZE_PX));
        checkCudaErrors(hipDeviceSynchronize());

        *d_index = 0;
        memset(d_image, 0, IMAGE_SIZE_PX * IMAGE_SIZE_PX);

        fillTiles<TILE_COUNT> <<< grid, threads >>>(d_out, d_image, d_index);
        const auto code = hipGetLastError();
        if (code != hipSuccess) {
            fprintf(stderr, "Failed to launch: %s\n",
                    hipGetErrorString(code));
        }
        checkCudaErrors(hipDeviceSynchronize());
        for (unsigned i=0; i < IMAGE_SIZE_PX * IMAGE_SIZE_PX; ++i) {
            const char c = d_image[i] ? ('0' + (i%10)) : ' ';
            printf("%c", c);
            if (i && !(i % IMAGE_SIZE_PX)) {
                printf("\n");
            }
        }
    }
    return d_out;
}

/**
 * Program main
 */
int main(int argc, char **argv)
{
    auto X = libfive::Tree::X();
    auto Y = libfive::Tree::Y();
    auto circle = sqrt(X*X + Y*Y) - 1.0;
    auto tape = prepareTape(circle);

    auto d_out = callProcessTiles(tape);
    hipDeviceSynchronize();
    Output out {0};
    checkCudaErrors(hipMemcpy(&out, d_out, sizeof(Output), hipMemcpyDeviceToHost));
    printf("%u %u\n", out.num_active, out.num_filled);

    return 0;
}
