#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <libfive/tree/opcode.hpp>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Our Interval arithmetic class
#include "gpu_interval.hpp"

struct Clause {
    const uint8_t opcode;
    const uint8_t banks;
    const uint16_t out;
    const uint16_t lhs;
    const uint16_t rhs;
};

struct Tape {
    const Clause* const __restrict__ tape;
    const uint32_t tape_length;
    const float* const __restrict__ constants;
};

__device__ void walk(const Tape tape,
                     const Interval X, const Interval Y,
                     Interval* const __restrict__ regs,
                     uint8_t* const __restrict__ choices)
{
    uint32_t choice_index = 0;
    for (uint32_t i=0; i < tape.tape_length; ++i) {
        const Clause c = tape.tape[i];
#define LHS (((c.banks & 1) ? regs[c.lhs] : Interval{tape.constants[c.lhs], \
                                                     tape.constants[c.lhs]}))
#define RHS (((c.banks & 1) ? regs[c.rhs] : Interval{tape.constants[c.rhs], \
                                                     tape.constants[c.rhs]}))
        using namespace libfive::Opcode;
        switch (c.opcode) {
            case VAR_X: regs[c.out] = X; break;
            case VAR_Y: regs[c.out] = Y; break;

            case OP_SQUARE: regs[c.out] = LHS.square(); break;
            case OP_SQRT: regs[c.out] = LHS.sqrt(); break;
            case OP_NEG: regs[c.out] = -LHS; break;
            // Skipping transcendental functions for now

            case OP_ADD: regs[c.out] = LHS + RHS; break;
            case OP_MUL: regs[c.out] = LHS * RHS; break;
            case OP_MIN: if (LHS.upper < RHS.lower) {
                             choices[choice_index] = 1;
                             regs[c.out] = LHS;
                         } else if (RHS.upper < LHS.lower) {
                             choices[choice_index] = 2;
                             regs[c.out] = RHS;
                         } else {
                             choices[choice_index] = 0;
                             regs[c.out] = LHS.min(RHS);
                         }
                         choice_index++;
                         break;
            case OP_MAX: if (LHS.lower > RHS.upper) {
                             choices[choice_index] = 1;
                             regs[c.out] = LHS;
                         } else if (RHS.lower > LHS.upper) {
                             choices[choice_index] = 2;
                             regs[c.out] = RHS;
                         } else {
                             choices[choice_index] = 0;
                             regs[c.out] = LHS.max(RHS);
                         }
                         choice_index++;
                         break;
            case OP_SUB: regs[c.out] = LHS - RHS; break;

            // Skipping various hard functions here
            default: break;
        }
    }
#undef LHS
#undef RHS
}

struct Output {
    uint32_t* const __restrict__ tiles;
    const uint32_t tiles_length;

    uint32_t num_active;
    uint32_t num_filled;
};

const static uint32_t IMAGE_SIZE_PX = 256;
const static uint32_t TILE_SIZE_PX = 16;
const static uint32_t TILE_COUNT = IMAGE_SIZE_PX / TILE_SIZE_PX;

const static uint32_t NUM_BLOCKS = 16;
const static uint32_t THREADS_PER_BLOCK = TILE_COUNT / NUM_BLOCKS;

__global__ void processTiles(const Tape tape,
        // Flat array for all pseudoregisters
        Interval* const __restrict__ regs_,
        const uint32_t num_regs,

        // Flat array for all CSG choices
        uint8_t* const __restrict__ csg_choices_,
        const uint32_t num_csg_choices,

        // Output data
        Output* const __restrict__ out)
{
    const float x = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    const float y = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;

    const Interval X = {x / TILE_COUNT, (x + 1) / TILE_COUNT};
    const Interval Y = {y / TILE_COUNT, (y + 1) / TILE_COUNT};

    // Unpack a 1D offset into the data arrays
    const uint32_t index = x * TILE_COUNT + y;
    Interval* __restrict__ const regs = regs_ + index * num_regs;
    uint8_t* __restrict__ const csg_choices = csg_choices_ + index * num_csg_choices;
    walk(tape, X, Y, regs, csg_choices);

    const Interval result = regs[tape.tape[tape.tape_length - 1].out];
    printf("[%f %f][%f %f]: [%f %f]\n",
            X.lower, X.upper,
            Y.lower, Y.upper,
            result.lower, result.upper);
}

/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
                                                        float *B, int wA,
                                                        int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
            a <= aEnd;
            a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16) {
        MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    } else {
        MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));

    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++) {
        if (block_size == 16) {
            MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        } else {
            MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                               static_cast<double>(dimsA.y) *
                               static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
                       (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
        " WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    printf("\nNOTE: The CUDA Samples are not meant for performance"\
           "measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv) {

    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 grid(NUM_BLOCKS, NUM_BLOCKS);

    {   // CUDA, help me pick magic numbers:
        int min_grid_size;
        int block_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
            processTiles);
        printf("Min grid size: %i\tBlock size: %i\n", min_grid_size, block_size);
    }

    processTiles <<< grid, threads >>>(Tape {},
        nullptr /* regs */, 0 /* num_regs */,
        nullptr /* csg_choices */, 0 /* num_csg_choices */,
        nullptr /* out */);

    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices" \
               " must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    int block_size = 32;

    dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
    dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y) {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
                                               dimsB.x, dimsB.y);

    int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}


